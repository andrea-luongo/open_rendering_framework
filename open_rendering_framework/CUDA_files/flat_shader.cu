#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_math.h>
#include "../structs.h"
#include "../LightSampler.h"
#include "../Microfacet.h"
using namespace optix;

// Standard ray variables
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
// Shadow variables
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_shadower, , );
//rtDeclareVariable(unsigned int, shadow_ray_type, , );
rtDeclareVariable(int, max_depth, , );

rtDeclareVariable(float3, diffuse_color, , );
rtDeclareVariable(float3, highlight_color, , );
rtDeclareVariable(uint, flat_shadow, , );
rtDeclareVariable(uint, highlight, , );
rtDeclareVariable(float, shininess, , );
rtDeclareVariable(float2, roughness, , );
rtDeclareVariable(uint, normal_distribution, , );
rtDeclareVariable(float, ior, , );
rtDeclareVariable(float, highlight_threshold, , );


// Any hit program for shadows
RT_PROGRAM void any_hit()
{
	// this material is opaque, so it fully attenuates all shadow rays
	prd_shadow.attenuation = 0.0f;
	rtTerminateRay();
}

// Closest hit program for Lambertian shading using the basic light as a directional source.
// This one includes shadows.
RT_PROGRAM void closest_hit()
{

	if (prd_radiance.depth > max_depth) return;
	float3 hit_pos = ray.origin + t_hit * ray.direction;
	float3 normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 ffnormal = faceforward(normal, -ray.direction, normal);
	float3 w_i = -ray.direction;
	uint& t = prd_radiance.seed;
	float3 result = diffuse_color ;

	//sampling microfacet normal
	float3 microfacet_normal;
	float z1 = rnd_tea(t);
	float z2 = rnd_tea(t);
	float n1_over_n2 = 1.0f / ior;
	microfacet_sample_visible_normal(w_i, ffnormal, microfacet_normal, roughness.x, roughness.y, z1, z2, normal_distribution);


	//sample light
	uint light_idx = light_buffer.size()*rnd_tea(t);
	float light_pdf = 1.0f / light_buffer.size();
	LightStruct direct_light = light_buffer[light_idx];
	float dist;
	float3 radiance;
	float3 w_l = make_float3(0.0f);
	evaluate_direct_illumination(hit_pos, &direct_light, w_l, radiance, dist, prd_radiance.seed);
	radiance /= light_pdf;

	float cos_theta = dot(ffnormal, w_l);

	float V = 0.0f;
	if (cos_theta > 0.0)
	{
		V = 1.0f;
		PerRayData_shadow shadow_prd;
		shadow_prd.attenuation = 1.0f;
		Ray shadow_ray(hit_pos, w_l, shadow_ray_type, scene_epsilon, dist - scene_epsilon);
		rtTrace(top_shadower, shadow_ray, shadow_prd);
		V = shadow_prd.attenuation;

	}

	if (flat_shadow == 1) {
		
			result *= V;
		
	}
	if (highlight == 1 && V > 0.0f) {

		float3 refl_normal = normalize(w_i + w_l);
		float cos_theta_refl = dot(refl_normal, w_l);
		//// Compute Fresnel reflectance (R) 
		float R_refl = 1.0f;
		float sin_theta_refl_t_sqr = n1_over_n2*n1_over_n2*(1.0f - cos_theta_refl*cos_theta_refl);
		float cos_theta_refl_t = 0;
		if (sin_theta_refl_t_sqr < 1.0f)
		{
			cos_theta_refl_t = sqrtf(1.0f - sin_theta_refl_t_sqr);
			R_refl = fresnel_R(cos_theta_refl, cos_theta_refl_t, n1_over_n2);
		}
		float T_01_refl = 1.0f - R_refl;


		float G_i_m_refl = masking_G1(w_i, refl_normal, ffnormal, roughness.x, roughness.y, normal_distribution);
		float G_o_m_refl = masking_G1(w_l, refl_normal, ffnormal, roughness.x, roughness.y, normal_distribution);
		float D_refl = microfacet_eval_visible_normal(w_i, refl_normal, ffnormal,roughness.x, roughness.y, normal_distribution);
		float weight_refl = G_i_m_refl * G_o_m_refl * D_refl * R_refl / (4.0f * fabsf(dot(w_i, ffnormal) * dot(w_l, ffnormal)));
	
		
		//MyComplex eta = MyComplex{ n1_over_n2, 0 };
		//MyComplex3 eta3 = { eta, eta, eta };
		//float3 weight_refl = make_float3(0.0f);
		//weight_refl = microfacet_multiscattering_conductor_BSDF_eval(w_i, w_l, ffnormal, eta3, roughness.x, roughness.y, t, 0, normal_distribution);
	

		float3 specular_contribution = radiance * weight_refl;
		float3 luminance_weight = make_float3(0.2126f, 0.7152f, 0.0722f);
		float luminance = dot(luminance_weight, specular_contribution);
		//float3 specular_contribution = highlight_color * fmaxf(0.0f, powf(dot(refl_normal, microfacet_normal), shininess)) * V;
		//if (luminance.x > highlight_threshold || luminance.y>highlight_threshold || luminance.z > highlight_threshold)
		if (luminance > highlight_threshold)
			//result += specular_contribution *V*highlight_color;
			result = highlight_color *V;
	}

	prd_radiance.result = result;
}
