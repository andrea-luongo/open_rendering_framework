#include "hip/hip_runtime.h"
// 02576 OptiX Rendering Framework
// Written by Jeppe Revall Frisvad, 2011
// Copyright (c) DTU Informatics 2011

#include <optix_world.h>
#include "../structs.h"

using namespace optix;

// Standard ray variables
rtDeclareVariable(PerRayData_depth, prd_depth, rtPayload, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
//
//// Variables for shading
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(rtObject, top_object, , );


// Closest hit program for drawing shading normals
RT_PROGRAM void closest_hit()
{
	float3 normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 ffnormal = faceforward(normal, -ray.direction, normal);

	prd_depth.normal= ffnormal;
	prd_depth.ray_depth = t_hit;

}