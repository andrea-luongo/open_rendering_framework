#include "hip/hip_runtime.h"
// 02576 OptiX Rendering Framework
// Written by Jeppe Revall Frisvad, 2011
// Copyright (c) DTU Informatics 2011

#include <optix_world.h>
#include "../structs.h"
#include "../Fresnel.h"
#include "../Microfacet.h"
#include "../random.h"
using namespace optix;

// Standard ray variables
rtDeclareVariable(PerRayData_depth, prd_depth, rtPayload, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
//
//// Variables for shading
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(rtObject, top_object, , );

rtDeclareVariable(uint, normal_distribution, , );
rtDeclareVariable(uint, microfacet_model, , );
rtDeclareVariable(float2, roughness, , );

// Closest hit program for drawing shading normals
RT_PROGRAM void closest_hit()
{
	float3 normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 ffnormal = faceforward(normal, -ray.direction, normal);
	uint& t = prd_depth.seed;
	float3 w_i = -ray.direction;
	float3 microfacet_normal;
	float a_x = roughness.x;
	float a_y = roughness.y;
	float z1 = rnd_tea(t);
	float z2 = rnd_tea(t);
	microfacet_sample_visible_normal(w_i, ffnormal, microfacet_normal, a_x, a_y, z1, z2, normal_distribution);

	prd_depth.normal= microfacet_normal;
	prd_depth.ray_depth = t_hit;

}