#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "envmap.h"
#include "structs.h"
#include "sampler.h"

using namespace optix;

#define INDIRECT

// Standard ray variables
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );

// Variables for shading
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, texcoord, attribute texcoord, ); 

// Material properties (corresponding to OBJ mtl params)
rtDeclareVariable(float3, emissive, , );
rtTextureSampler<float4, 2> diffuse_map; 

// Shadow variables
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_shadower, , );
rtDeclareVariable(unsigned int, shadow_ray_type, , );

#ifdef INDIRECT
// Recursive ray tracing variables
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(int, max_depth, , );
#endif

// Any hit program for shadows
RT_PROGRAM void any_hit_shadow()
{
  // this material is opaque, so it fully attenuates all shadow rays
  prd_shadow.attenuation = 0.0f;
  rtTerminateRay();
}

// Closest hit program for Lambertian shading using a triangle mesh as an area source.
// This one includes shadows.
RT_PROGRAM void envmap_shader() 
{ 
  float3 hit_pos = ray.origin + t_hit*ray.direction; 
  float3 normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal)); 
  float3 ffnormal = faceforward(normal, -ray.direction, normal); 
  float3 rho_d = make_float3(tex2D(diffuse_map, texcoord.x, texcoord.y));
  uint& t = prd_radiance.seed;
  float3 color = prd_radiance.emit ? emissive : make_float3(0.0f);
  
  // Direct illumination
  float3 w_i;
  float3 L_i;
  sample_environment(hit_pos, w_i, L_i, t);

  float cos_theta = dot(ffnormal, w_i);
  if(cos_theta > 0.0)
  { 
    PerRayData_shadow shadow_prd;
    shadow_prd.attenuation = 1.0f;
    Ray shadow_ray = make_Ray(hit_pos, w_i, shadow_ray_type, scene_epsilon, RT_DEFAULT_MAX);
    rtTrace(top_shadower, shadow_ray, shadow_prd);
    float V = shadow_prd.attenuation;
    color += V*L_i*rho_d*M_1_PIf*cos_theta;
  }
#ifdef INDIRECT
  // Indirect illumination
  float prob = (rho_d.x + rho_d.y + rho_d.z)/3.0f;
  if(rnd(t) < prob && prd_radiance.depth < max_depth)
  {
    float3 new_dir = sample_cosine_weighted(ffnormal, t);
    ++prd_radiance.depth;
    prd_radiance.emit = 0;
    Ray new_ray(hit_pos, new_dir, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
    rtTrace(top_object, new_ray, prd_radiance);
    color += rho_d*prd_radiance.result/prob;
  }
#endif
  prd_radiance.result = color; 
}
