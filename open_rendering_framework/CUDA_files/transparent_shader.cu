#include "hip/hip_runtime.h"
// 02576 OptiX Rendering Framework
// Written by Jeppe Revall Frisvad, 2011
// Copyright (c) DTU Informatics 2011

#include <optix.h>
#include <optix_math.h>
#include "../random.h"
#include "../structs.h"
#include "../sampler.h"
#include "../LightSampler.h"
#include "../Fresnel.h"
//#define DIFFUSE_PART

using namespace optix;

// Standard ray variables
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(int, max_depth, , );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

// Variables for shading
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );

// Material properties (corresponding to OBJ mtl params)
rtTextureSampler<float4, 2> diffuse_map;
rtDeclareVariable(float, ior, , );
rtDeclareVariable(float3, glass_absorption, , );

// Recursive ray tracing variables
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );
//rtDeclareVariable(unsigned int, radiance_ray_type, , );

// Any hit program for shadows
RT_PROGRAM void any_hit()
{
	// this material is opaque, so it fully attenuates all shadow rays
	prd_shadow.attenuation = 0.0f;
	rtTerminateRay();
}


// Closest hit program for drawing shading normals
RT_PROGRAM void closest_hit()
{
  prd_radiance.result = make_float3(0.0f);
  if(prd_radiance.depth > max_depth) return;
  ++prd_radiance.depth;
  // Initialize variables
  float3 hit_pos = ray.origin + t_hit*ray.direction; 
  float3 normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal)); 
  uint& t = prd_radiance.seed;

#ifdef DIFFUSE_PART
  // Diffuse part
  float3 ffnormal = faceforward(normal, -ray.direction, normal);
  float3 rho_d = make_float3(tex2D(diffuse_map, texcoord.x, texcoord.y));
  float prob_d = (rho_d.x + rho_d.y + rho_d.z)/3.0f;
  if(rnd_tea(t) < prob_d)
  {
    float3 new_dir = sample_cosine_weighted(ffnormal, t);
    ++prd_radiance.depth;
    prd_radiance.emit = 0;
    Ray new_ray(hit_pos, new_dir, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
    rtTrace(top_object, new_ray, prd_radiance);
    prd_radiance.result *= rho_d/prob_d;
    return;
  }
#endif

  // Russian roulette with absorption if inside
  float3 beam_T = make_float3(1.0f);
  float n1_over_n2 = 1.0f/ior;
  float cos_theta = dot(-ray.direction, normal);
  if(cos_theta < 0.0f)
  {
    beam_T = expf(-t_hit*glass_absorption);
    float prob = (beam_T.x + beam_T.y + beam_T.z)/3.0f;
    if(rnd_tea(t) >= prob) return;
    beam_T /= prob;
    n1_over_n2 = ior;
    normal = -normal;
    cos_theta = -cos_theta;
  }

  //// Compute Fresnel reflectance (R) and trace compute reflected and refracted directions
  float R = 1.0f;
  float sin_theta_t_sqr = n1_over_n2*n1_over_n2*(1.0f - cos_theta*cos_theta);
  float cos_theta_t = 0;
  if(sin_theta_t_sqr < 1.0f)
  {
    cos_theta_t = sqrtf(1.0f - sin_theta_t_sqr);
    R = fresnel_R(cos_theta, cos_theta_t, n1_over_n2);
  }
  float reflect_xi = rnd_tea(t);


  //prd_radiance.emit = 1;
 
  //Russian Roulette to choose between reflection and refraction
  float3 dir;
  if(reflect_xi < R)
    dir = reflect(ray.direction, normal);
  else
    dir = n1_over_n2*ray.direction + normal*(n1_over_n2*cos_theta - cos_theta_t);

  prd_radiance.emit_light = 1;
  Ray to_trace = make_Ray(hit_pos, dir, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
  rtTrace(top_object, to_trace, prd_radiance);
  prd_radiance.result *= beam_T;
 
}
