#include "hip/hip_runtime.h"
// 02576 OptiX Rendering Framework
// Written by Jeppe Revall Frisvad, 2011
// Copyright (c) DTU Informatics 2011

#include <optix_world.h>
#include "../structs.h"

using namespace optix;

// Standard ray variables
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );
//
//// Variables for shading
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(rtObject, top_object, , );
// Any hit program for shadows
RT_PROGRAM void any_hit()
{
  //prd_shadow.attenuation = 0.0f;
  rtTerminateRay();
}

// Closest hit program for drawing shading normals
RT_PROGRAM void closest_hit()
{
	float3 normal = rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal);
	prd_radiance.result = normal*0.5f + 0.5f;
}