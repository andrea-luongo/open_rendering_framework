#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_math.h>
#include "../random.h"
#include "../structs.h"
#include "../sampler.h"
#include "../LightSampler.h"
#include "../Fresnel.h"
#include "../Microfacet.h"
#include "../MyComplex.h"
using namespace optix;



// Standard ray variables
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

// Variables for shading
//rtBuffer<LightStruct> light_buffer;
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(int, max_depth, , );

// Material properties 
rtTextureSampler<float4, 2> diffuse_map;
rtDeclareVariable(float3, emissive, , );
rtDeclareVariable(float3, diffuse_color, , );
rtDeclareVariable(float, ior, , );
rtDeclareVariable(uint, normal_distribution, , );
rtDeclareVariable(uint, microfacet_model, , );
rtDeclareVariable(float2, roughness, , );

// Shadow variables
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_shadower, , );
//rtDeclareVariable(unsigned int, shadow_ray_type, , );


// Recursive ray tracing variables
rtDeclareVariable(rtObject, top_object, , );
//rtDeclareVariable(unsigned int, radiance_ray_type, , );

// Any hit program for shadows
RT_PROGRAM void any_hit()
{
	// this material is opaque, so it fully attenuates all shadow rays
	prd_shadow.attenuation = 0.0f;
	rtTerminateRay();
}

// Closest hit program for Lambertian shading using the basic light as a directional source.
// This one includes shadows.
RT_PROGRAM void closest_hit()
{

	if (prd_radiance.depth > max_depth) return;

	float3 hit_pos = ray.origin + t_hit * ray.direction;
	float3 normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 ffnormal = faceforward(normal, -ray.direction, normal);
	float3 w_i = -ray.direction;
	// Emission
	float3 result = make_float3(0.0f);
	uint& t = prd_radiance.seed;

	//refraction properties
	float n1_over_n2 = 1.0f / ior;
	float r_10 = two_C1(ior);
	float t_10 = 1.0f - r_10;

	float a_x = roughness.x;
	float a_y = roughness.y;
	//float3 up_normal = make_float3(0.0f, 1.0f, 0.0f);
	//float roughness_scale_factor = fabsf(dot(up_normal, ffnormal));
	//roughness_scale_factor = powf(roughness_scale_factor, 1.0f/2.0f);
	//a_x = (1 - a_x) * roughness_scale_factor + a_x;
	//a_y = (1 - a_y) * roughness_scale_factor + a_y;

	//rtPrintf("scale %f \n", roughness_scale_factor);
	//sampling microfacet normal
	float3 microfacet_normal;
	float z1 = rnd_tea(t);
	float z2 = rnd_tea(t);
	microfacet_sample_visible_normal(w_i, ffnormal, microfacet_normal, a_x, a_y, z1, z2, normal_distribution);

	float cos_theta_i = dot(w_i, microfacet_normal);
	//// Compute Fresnel reflectance (R)
	float R_i = 1.0f;
	float sin_theta_t_sqr = n1_over_n2*n1_over_n2*(1.0f - cos_theta_i*cos_theta_i);
	float cos_theta_t = 0;
	if (sin_theta_t_sqr < 1.0f)
	{
		cos_theta_t = sqrtf(1.0f - sin_theta_t_sqr);
		R_i = fresnel_R(cos_theta_i, cos_theta_t, n1_over_n2);
	}

	float T_01_i = 1.0f - R_i;
	// Direct illumination

	uint light_idx = light_buffer.size()*rnd_tea(t);
	float light_pdf = 1.0f / light_buffer.size();
	LightStruct direct_light = light_buffer[light_idx];

	float dist;
	float3 radiance;
	float3 w_l = make_float3(0.0f);
	float cos_theta_l = 0.0f;
	evaluate_direct_illumination(hit_pos, &direct_light, w_l, radiance, dist, prd_radiance.seed);
	radiance /= light_pdf;
	cos_theta_l = dot(ffnormal, w_l);
	if (cos_theta_l > 0.0)
	{
		float V = 1.0f;
		PerRayData_shadow shadow_prd;
		shadow_prd.attenuation = 1.0f;
		Ray shadow_ray(hit_pos, w_l, shadow_ray_type, scene_epsilon, dist);
		rtTrace(top_shadower, shadow_ray, shadow_prd);
		V = shadow_prd.attenuation;
		if (V > 0.0f) 
		{
			
			float3 Li = V*radiance;

			if (microfacet_model == MULTISCATTERING_MODEL)
			{
				float3 diffuse_brdf = make_float3(0.0f);
				diffuse_brdf = microfacet_multiscattering_diffuse_BSDF_eval(w_i, w_l, ffnormal, a_x, a_y, t, 0, diffuse_color, normal_distribution);
				result += Li * diffuse_brdf ;
				
				MyComplex eta = MyComplex{ n1_over_n2, 0 };
				MyComplex3 eta3 = { eta, eta, eta };
				float3 reflected_brdf = make_float3(0.0f);
				reflected_brdf = microfacet_multiscattering_conductor_BSDF_eval(w_i, w_l, ffnormal, eta3, a_x, a_y, t, 0, normal_distribution);
				
				result += Li * reflected_brdf ;
				prd_radiance.emit_light = 0;
			}
			else {

				float3 refl_normal = normalize(w_i + w_l);
				float cos_theta_refl = dot(refl_normal, w_l);
				//// Compute Fresnel reflectance (R) 
				float R_refl = 1.0f;
				float sin_theta_refl_t_sqr = n1_over_n2*n1_over_n2*(1.0f - cos_theta_refl*cos_theta_refl);
				float cos_theta_refl_t = 0;
				if (sin_theta_refl_t_sqr < 1.0f)
				{
					cos_theta_refl_t = sqrtf(1.0f - sin_theta_refl_t_sqr);
					R_refl = fresnel_R(cos_theta_refl, cos_theta_refl_t, n1_over_n2);
				}
				float T_01_refl = 1.0f - R_refl;

				float cos_theta_diff = dot(microfacet_normal, w_l);
				float R_diff = 1.0f;
				float sin_theta_diff_t_sqr = n1_over_n2*n1_over_n2*(1.0f - cos_theta_diff*cos_theta_diff);
				float cos_theta_diff_t = 0;
				if (sin_theta_diff_t_sqr < 1.0f)
				{
					cos_theta_diff_t = sqrtf(1.0f - sin_theta_diff_t_sqr);
					R_diff = fresnel_R(cos_theta_diff, cos_theta_diff_t, n1_over_n2);
				}
				float T_01_diff = 1.0f - R_diff;

				float G_i_m_refl = masking_G1(w_i, refl_normal, ffnormal, a_x, a_y, normal_distribution);
				float G_o_m_refl = masking_G1(w_l, refl_normal, ffnormal, a_x, a_y, normal_distribution);
				float G_o_m_diff = masking_G1(w_l, microfacet_normal, ffnormal, a_x, a_y, normal_distribution);
				float D_refl = microfacet_eval_visible_normal(w_i, refl_normal, ffnormal, a_x, a_y, normal_distribution);
				float weight_refl = G_i_m_refl * G_o_m_refl * D_refl / (4.0f * fabsf(dot(w_i, ffnormal) * dot(w_l, ffnormal)));
				float weight_diff = G_o_m_diff;
			
				result += Li* M_1_PIf*cos_theta_l * diffuse_color * n1_over_n2 * n1_over_n2 * T_01_i * T_01_diff / (1.0f - diffuse_color * r_10) * weight_diff;
				result += Li* R_refl * weight_refl ;
				prd_radiance.emit_light = 0;
			}
		}
	}
	//}

	// Indirect illumination 

	float prob = R_i;
	float xi = rnd_tea(t);
	if (xi > prob)
	{
		if (microfacet_model == MULTISCATTERING_MODEL)
		{
			uint scatteringOrder = 0;
			float3 w_o;
			float3 w_m;
			float3 weight;
			microfacet_multiscattering_diffuse_BSDF_sample(w_i, w_o, w_m, ffnormal, a_x, a_y, t, scatteringOrder, weight, diffuse_color, normal_distribution);

			// Compute Fresnel reflectance (R) and compute diffuse contribution
			float cos_theta_r = dot(w_o, w_m);
			float R_r = 1.0f;
			float sin_theta_r_t_sqr = n1_over_n2*n1_over_n2*(1.0f - cos_theta_r*cos_theta_r);
			float cos_theta_r_t = 0;
			if (sin_theta_r_t_sqr < 1.0f)
			{
				cos_theta_r_t = sqrtf(1.0f - sin_theta_r_t_sqr);
				R_r = fresnel_R(cos_theta_r, cos_theta_r_t, n1_over_n2);
			}
			float T_01_r = 1.0f - R_r;
			
			PerRayData_radiance prd_diffuse;
			prd_diffuse.depth = prd_radiance.depth + 1;
			prd_diffuse.seed = t;
			prd_diffuse.emit_light = prd_radiance.emit_light;
			prd_diffuse.result = make_float3(0.0f);
			Ray diffuse_ray(hit_pos, w_o, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
			rtTrace(top_object, diffuse_ray, prd_diffuse);

			result += prd_diffuse.result * n1_over_n2 * n1_over_n2  * T_01_r / (1.0f - diffuse_color * r_10)  * weight;
			prd_radiance.seed = prd_diffuse.seed;

		}
		else {
			float3 diffuse_dir = sample_cosine_weighted(microfacet_normal, t);
			float cos_theta_r = dot(diffuse_dir, microfacet_normal);
			float G_o_m = masking_G1(diffuse_dir, microfacet_normal, ffnormal, a_x, a_y, normal_distribution);
			if (G_o_m < 0.0f) {
				prd_radiance.result = result;
				return;
			}
			// Compute Fresnel reflectance (R) and compute diffuse contribution
			float R_r = 1.0f;
			float sin_theta_r_t_sqr = n1_over_n2*n1_over_n2*(1.0f - cos_theta_r*cos_theta_r);
			float cos_theta_r_t = 0;
			if (sin_theta_r_t_sqr < 1.0f)
			{
				cos_theta_r_t = sqrtf(1.0f - sin_theta_r_t_sqr);
				R_r = fresnel_R(cos_theta_r, cos_theta_r_t, n1_over_n2);
			}
			float T_01_r = 1.0f - R_r;

			PerRayData_radiance prd_diffuse;
			prd_diffuse.depth = prd_radiance.depth + 1;
			prd_diffuse.seed = t;
			prd_diffuse.emit_light = prd_radiance.emit_light;
			prd_diffuse.result = make_float3(0.0f);
			Ray diffuse_ray(hit_pos, diffuse_dir, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
			rtTrace(top_object, diffuse_ray, prd_diffuse);

			result += prd_diffuse.result * diffuse_color * n1_over_n2 * n1_over_n2 *T_01_r / (1.0f - diffuse_color * r_10)  * G_o_m;
			prd_radiance.seed = prd_diffuse.seed;
		}
	}
	else 
	{
		if (microfacet_model == MULTISCATTERING_MODEL)
		{
			MyComplex eta = MyComplex{ n1_over_n2, 0 };
			MyComplex3 eta3 = { eta, eta, eta };
			uint scatteringOrder = 10;
			float3 w_o;
			float3 w_m;
			float3 weight;
			microfacet_multiscattering_conductor_BSDF_sample(w_i, w_o, w_m, normal, eta3, a_x, a_y, t, scatteringOrder, weight, normal_distribution);
			PerRayData_radiance prd_new_ray;
			prd_new_ray.depth = prd_radiance.depth + 1;
			prd_new_ray.result = make_float3(0.0f);
			prd_new_ray.seed = t;
			prd_new_ray.emit_light = prd_radiance.emit_light;
			optix::Ray new_ray = optix::make_Ray(hit_pos, w_o, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
			rtTrace(top_object, new_ray, prd_new_ray);
			result += prd_new_ray.result ;
		}
		else {
			//compute reflectance contribution
			float3 refl_dir = reflect(ray.direction, microfacet_normal);
			float G_o_m = masking_G1(refl_dir, microfacet_normal, ffnormal, a_x, a_y, normal_distribution);
			if (G_o_m < 0.0f) {
				prd_radiance.result = result;
				return;
			}
			PerRayData_radiance prd_refl;
			prd_refl.depth = prd_radiance.depth + 1;
			prd_refl.seed = prd_radiance.seed;
			prd_refl.emit_light = prd_radiance.emit_light;
			prd_refl.result = make_float3(0.0f);
			Ray refl_ray(hit_pos, refl_dir, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
			rtTrace(top_object, refl_ray, prd_refl);
			result += prd_refl.result * G_o_m;
			prd_radiance.seed = prd_refl.seed;
		}
	}

	//prd_radiance.depth++;
	prd_radiance.result = result;
}
