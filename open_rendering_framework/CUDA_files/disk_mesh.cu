#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include "../helpers.h"
using namespace optix;

// This is to be plugged into an RTgeometry object to represent
// a triangle mesh with a vertex buffer of triangle soup (triangle list)
// with an interleaved position, normal, texturecoordinate layout.

rtDeclareVariable(float3, radiance, , );
rtDeclareVariable(float3, position, , );
rtDeclareVariable(float3, normal, , );
rtDeclareVariable(float, radius, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, scene_epsilon, , );
RT_PROGRAM void disk_intersect(int primIdx)
{

	float3 O = ray.origin - position;
	
	float den = dot(normal, ray.direction);
	if (fabsf(den) > scene_epsilon) {
		
		float t = -dot(O, normal) / den;
		//if(t > ray.tmin && t < ray.tmax){
		float3 point = ray.origin + t*ray.direction;
		float3 v = point - position;
		float square_dist = dot(v, v);
		if (square_dist <= radius*radius) {
			if (rtPotentialIntersection(t)) {
				shading_normal = geometric_normal = normal;
				
				rtReportIntersection(0);
			}
		}
	}

}

RT_PROGRAM void disk_bounds(int , float result[6])
{

	optix::Aabb* aabb = (optix::Aabb*)result;
	float3 extent = radius * sqrt(1.0f - normal*normal);
	if (radius > 0.0f && !isinf(radius)) {
		aabb->m_min = position - extent;
		aabb->m_max = position + extent;
	}
	else {
		aabb->invalidate();
	}
}

