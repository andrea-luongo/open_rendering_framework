#include "hip/hip_runtime.h"
// 02576 OptiX Rendering Framework
// Written by Jeppe Revall Frisvad, 2011
// Copyright (c) DTU Informatics 2011

#include <optix.h>
#include <optix_math.h>
#include "../structs.h"
#include "../Microfacet.h"
#include "../MyComplex.h"
#include "../random.h"


using namespace optix;
// Standard ray variables
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(int, max_depth, , );

// Variables for shading
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

// Recursive ray tracing variables
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(rtObject, top_shadower, , );
//rtDeclareVariable(unsigned int, shadow_ray_type, , );
//rtDeclareVariable(unsigned int, radiance_ray_type, , );

// Material properties 
rtDeclareVariable(MyComplex3, ior, , );
rtDeclareVariable(float2, roughness, , );
rtDeclareVariable(uint, microfacet_model, , );
rtDeclareVariable(uint, normal_distribution, , );

// Any hit program for shadows
RT_PROGRAM void any_hit()
{
	// this material is opaque, so it fully attenuates all shadow rays
	prd_shadow.attenuation = 0.0f;
	rtTerminateRay();
}
// Closest hit program for drawing shading normals
RT_PROGRAM void closest_hit()
{
	if (prd_radiance.depth >= max_depth)
	{
		prd_radiance.result = make_float3(0.0f);
		return;
	}
	float3 result = make_float3(0.0f);
	// Compute cosine to angle of incidence
	float3 hit_point = ray.origin + t_hit * ray.direction;
	float3 normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 ffnormal = faceforward(normal, -ray.direction, normal);
	float3 w_i = -ray.direction;

	float n1 = 1.0f;
	
	MyComplex3 eta = n1 / ior;
	uint& seed = prd_radiance.seed;

	float a_x = roughness.x;
	float a_y = roughness.y;
	if (microfacet_model == MULTISCATTERING_MODEL)
	{
		uint scatteringOrder = 0;
		float3 w_o;
		float3 w_m;
		float3 weight;
		microfacet_multiscattering_conductor_BSDF_sample(w_i, w_o, w_m, normal, eta, a_x, a_y, seed, scatteringOrder, weight, normal_distribution);
		PerRayData_radiance prd_new_ray;
		prd_new_ray.depth = prd_radiance.depth + 1;
		prd_new_ray.result = make_float3(0.0f);
		prd_new_ray.seed = seed;
		prd_new_ray.emit_light = 1;
		optix::Ray new_ray = optix::make_Ray(hit_point, w_o, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
		rtTrace(top_object, new_ray, prd_new_ray);

		result += prd_new_ray.result * weight;

	}
	else
	{
		float z1 = rnd_tea(seed);
		float z2 = rnd_tea(seed);
		float3 microfacet_normal;
		float G_i_m;
		float G_o_m_refl;
		float G_o_m_refr;
		if (microfacet_model == WALTER_MODEL)
		{
			microfacet_sample_normal(ffnormal, microfacet_normal, z1, z2, a_x, normal_distribution);
		}
		else if (microfacet_model == VISIBLE_NORMALS_MODEL)
		{
			microfacet_sample_visible_normal(w_i, ffnormal, microfacet_normal, a_x, a_y, z1, z2, normal_distribution);
		}


		if (microfacet_model == WALTER_MODEL)
		{
			G_i_m = masking_G1(w_i, microfacet_normal, ffnormal, a_x, normal_distribution);
		}
		else if (microfacet_model == VISIBLE_NORMALS_MODEL)
		{
			G_i_m = masking_G1(w_i, microfacet_normal, ffnormal, a_x, a_y, normal_distribution);
		}

		//stop ray tracing if the microfacet is not visible from direction i
		if (G_i_m <= 0.0f) {
			prd_radiance.result = result;
			return;
		}
		// Compute Fresnel reflectance (R) and reflected and refracted rays
		float3 F = fresnel_MyComplex_R(w_i, microfacet_normal, eta);
		float3 w_o =  -w_i + 2.0f*microfacet_normal*dot(w_i, microfacet_normal);

		float abs_i_m = fabsf(dot(w_i, microfacet_normal));
		float abs_i_n = fabsf(dot(w_i, ffnormal));
		float abs_n_m = fabsf(dot(normal, microfacet_normal));
		PerRayData_radiance prd_new_ray;
		prd_new_ray.depth = prd_radiance.depth + 1;
		prd_new_ray.result = make_float3(0.0f);
		prd_new_ray.seed = seed;
		prd_new_ray.emit_light = 1;
		optix::Ray reflected_ray = optix::make_Ray(hit_point, w_o, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
		float3 weight = F;
		{
			// Trace reflected ray
			if (microfacet_model == WALTER_MODEL)
			{
				G_o_m_refl = masking_G1(w_o, microfacet_normal, ffnormal, a_x, normal_distribution);
			}
			else if (microfacet_model == VISIBLE_NORMALS_MODEL)
			{
				G_o_m_refl = masking_G1(w_o, microfacet_normal, ffnormal, a_x, a_y, normal_distribution);
			}

			if (G_o_m_refl <= 0.0f) {
				prd_radiance.result = result;
				return;
			}
			rtTrace(top_object, reflected_ray, prd_new_ray);
			if (microfacet_model == WALTER_MODEL)
			{
				weight *= abs_i_m * G_i_m * G_o_m_refl / (abs_i_n * abs_n_m);
			}
			else if (microfacet_model == VISIBLE_NORMALS_MODEL)
			{
				weight *= G_o_m_refl;
			}

		}
		prd_radiance.seed = prd_new_ray.seed;
		result += prd_new_ray.result * weight;
	}

	prd_radiance.result = result;

}