#include "hip/hip_runtime.h"
// 02576 OptiX Rendering Framework
// Written by Jeppe Revall Frisvad, 2011
// Copyright (c) DTU Informatics 2011

#include <optix_world.h>
#include "../structs.h"
#include "../random.h"

using namespace optix;

// Camera variables
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
//
// Ray generation variables
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );
//rtDeclareVariable(uint, radiance_ray_type, , );
rtDeclareVariable(uint, frame, , );
rtDeclareVariable(uint2, patch_origin, , );
rtDeclareVariable(uint2, patch_dims, , );
// Window variables
rtBuffer<float4, 2> output_buffer;

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );
// Exception and debugging variables
rtDeclareVariable(float3, exception_color, , );

RT_PROGRAM void path_tracer()
{
	float3 result = make_float3(0.0f);

	if (launch_index.x < patch_origin.x || launch_index.x > patch_origin.x + patch_dims.x) {
		output_buffer[launch_index] = (make_float4(result, 1.0f)) ;
		return;
	}
	if(launch_index.y < patch_origin.y || launch_index.y > patch_origin.y+ patch_dims.y) {
		output_buffer[launch_index] = (make_float4(result, 1.0f));
		return;
	}

	PerRayData_radiance prd;
	prd.emit_light = 1;
	prd.depth = 0;
	prd.seed = tea<16>(launch_dim.x*launch_index.y + launch_index.x, frame);
	prd.seed64.seed = make_uint2(tea<16>(launch_dim.x*launch_index.y + launch_index.x, frame), tea<16>(launch_dim.x*launch_index.y + launch_index.x, frame));
	/*prd.seed64.l = tea<16>(launch_dim.x*launch_index.y + launch_index.x, frame);*/
	float2 jitter = make_float2(rnd_tea(prd.seed), rnd_tea(prd.seed));
	float2 ip_coords = (make_float2(launch_index) + jitter) / make_float2(launch_dim) * 2.0f - 1.0f;
	float3 origin = eye;
	float3 direction = normalize(ip_coords.x*U + ip_coords.y*V + W);
	Ray ray(origin, direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);

	prd.result = make_float3(0.0f);
;

	rtTrace(top_object, ray, prd);
	result += prd.result;


	if (isfinite(result.x) && isfinite(result.y) && isfinite(result.z))
	{
		float4 curr_sum = (frame != 0) ? output_buffer[launch_index] * ((float)frame) : make_float4(0.0f);
		output_buffer[launch_index] = (make_float4(result, 1.0f) + curr_sum) / ((float)(frame + 1));
	
	}
	//if (launch_index.x == launch_dim.x/2 && launch_index.y == launch_dim.y / 2)
	//	output_buffer[launch_index] = (make_float4(0.0f, 100.0f, 0.0f, 1.0f));
	/*float4 curr_sum = (frame != 0) ? output_buffer[launch_index] * ((float)frame) : make_float4(0.0f);
	output_buffer[launch_index] = (make_float4(result, 0.0f) + curr_sum) / ((float)(frame + 1));*/
}

RT_PROGRAM void exception()
{
	output_buffer[launch_index] = make_float4(exception_color, 1.0f);

}
