#include "hip/hip_runtime.h"
// 02576 OptiX Rendering Framework
// Written by Jeppe Revall Frisvad, 2011
// Copyright (c) DTU Informatics 2011

#include <optix_world.h>
#include <optix.h>
#include <optix_math.h>
#include "../helpers.h"
#include "../random.h"
#include "../structs.h"
#include "../sampler.h"
#include "../Fresnel.h"
#include "../LightSampler.h"
#include "../Microfacet.h"
using namespace optix;
#define GLOBAL
#define RND_64

// Triangle mesh data
rtBuffer<float3> vertex_buffer;
rtBuffer<float3> normal_buffer;
rtBuffer<int3>   vindex_buffer;
rtBuffer<int3>   nindex_buffer;

// Ray generation variables
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_shadower, , );
//rtDeclareVariable(unsigned int, shadow_ray_type, , );
#ifdef GLOBAL
rtDeclareVariable(rtObject, top_object, , );
#endif
// SS properties
rtDeclareVariable(ScatteringMaterialProperties, current_scattering_properties, , );
rtDeclareVariable(float2, current_roughness, , );
rtDeclareVariable(uint, current_microfacet_model, , );
rtDeclareVariable(uint, current_normal_distribution, , );
rtDeclareVariable(uint, current_material_type, , );

// Window variables
rtBuffer<PositionSample> samples_output_buffer;
rtDeclareVariable(uint, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint, current_translucent_obj, , );
rtDeclareVariable(uint, samples, , );
rtDeclareVariable(uint, frame, , );
rtDeclareVariable(Matrix4x4, transform_matrix, , );
rtDeclareVariable(Matrix4x4, normal_matrix, , );

RT_PROGRAM void sample_camera()
{
	uint idx = launch_index + current_translucent_obj*samples;
	PositionSample& sample = samples_output_buffer[idx];

	uint triangles = vindex_buffer.size();
	uint t = tea<16>(idx, frame);
#ifdef RND_64
	Seed64 t64;
	t64.seed = make_uint2(tea<16>(idx, frame), tea<16>(idx, frame));
	uint triangle_id = (int)(rnd_accurate(t64) * triangles);
#else
	uint triangle_id = (int)(rnd_tea(t) * triangles);
#endif

	int3 idx_vxt = vindex_buffer[triangle_id];
	float3 v0 = vertex_buffer[idx_vxt.x];
	float3 v1 = vertex_buffer[idx_vxt.y];
	float3 v2 = vertex_buffer[idx_vxt.z];

	v0 = make_float3(transform_matrix * optix::make_float4(v0, 1.0f));
	v1 = make_float3(transform_matrix * optix::make_float4(v1, 1.0f));
	v2 = make_float3(transform_matrix * optix::make_float4(v2, 1.0f));

	float3 perp_triangle = cross(v1 - v0, v2 - v0);
	float area = 0.5f*length(perp_triangle);
	// sample a point in the triangle

#ifdef RND_64
	float xi1 = sqrt(rnd_accurate(t64));
	float xi2 = rnd_accurate(t64);
#else
	float xi1 = sqrt(rnd_tea(t));
	float xi2 = rnd_tea(t);
#endif
	float u = 1.0f - xi1;
	float v = (1.0f - xi2)*xi1;
	float w = xi1*xi2;
	float3 pos = u*v0 + v*v1 + w*v2;
	sample.pos = u*v0 + v*v1 + w*v2;
	//sample.pos = make_float3(transform_matrix * optix::make_float4(pos, 1.0f));
	float3 n;
	// compute the sample normal
	if (normal_buffer.size() > 0)
	{
		int3 nidx_vxt = nindex_buffer[triangle_id];
		float3 n0 = normal_buffer[nidx_vxt.x];
		float3 n1 = normal_buffer[nidx_vxt.y];
		float3 n2 = normal_buffer[nidx_vxt.z];
		n = normalize(u*n0 + v*n1 + w*n2);
		n = make_float3(normal_matrix * optix::make_float4(n, 0.0f));
		n = normalize(n);
	}
	else {
		n = normalize(perp_triangle);
	}

	sample.normal = n;

	float ior = current_scattering_properties.relative_ior;
	float recip_ior = 1.0f / ior;
	// evaluate incoming light

	float3 Le, w_i;
	float r;
	float cos_theta_i;
	float normal_pdf = 1.0f;
#ifdef GLOBAL 
	float indirect_prob = 0.5f;
	if (rnd(t) < indirect_prob)
	{
#endif

#ifdef RND_64 
	uint light_idx = light_buffer.size()*rnd_accurate(t64);
#else
	uint light_idx = light_buffer.size()*rnd_tea(t);
#endif
	float light_pdf = 1.0f / light_buffer.size();
	LightStruct direct_light = light_buffer[light_idx];
	evaluate_direct_illumination(sample.pos, &direct_light, w_i, Le, r, t);
	sample.dir = w_i;

	if (current_material_type != TRANSLUCENT_SHADER)
	{
		float3 microfacet_normal;
#ifdef RND_64
		float z1 = rnd_accurate(t64);
		float z2 = rnd_accurate(t64);
#else
		float z1 = rnd_tea(t);
		float z2 = rnd_tea(t);
#endif
		float a_x = current_roughness.x;
		float a_y = current_roughness.y;
		if (current_microfacet_model == WALTER_MODEL)
		{
			microfacet_sample_normal(n, microfacet_normal, a_x, z1, z2, current_normal_distribution);
		}
		else
		{
			microfacet_sample_visible_normal(w_i, n, microfacet_normal, a_x, a_y, z1, z2, current_normal_distribution);

		}
		sample.normal = microfacet_normal;
	}

	cos_theta_i = max(dot(w_i, n), 0.0f);
	if (cos_theta_i > 0.0)
	{
		PerRayData_shadow shadow_prd;
		shadow_prd.attenuation = 1.0f;
		Ray shadow_ray(sample.pos, w_i, shadow_ray_type, scene_epsilon, r);
		rtTrace(top_shadower, shadow_ray, shadow_prd);
		Le *= shadow_prd.attenuation* cos_theta_i / light_pdf;
	}
	else
	{
		Le = make_float3(0.0f);
	}
#ifdef GLOBAL
	}
	else
	{
		w_i = sample_cosine_weighted(n, t);

		if (current_material_type != TRANSLUCENT_SHADER)
		{
			float3 microfacet_normal;
#ifdef RND_64
			float z1 = rnd_accurate(t64);
			float z2 = rnd_accurate(t64);
#else
			float z1 = rnd_tea(t);
			float z2 = rnd_tea(t);
#endif
			float a_x = current_roughness.x;
			float a_y = current_roughness.y;
			if (current_microfacet_model == WALTER_MODEL)
			{
				microfacet_sample_normal(n, microfacet_normal, a_x, z1, z2, current_normal_distribution);
			}
			else
			{
				microfacet_sample_visible_normal(w_i, n, microfacet_normal, a_x, a_y, z1, z2, current_normal_distribution);
			}
			sample.normal = microfacet_normal;
		}

		PerRayData_radiance prd_new;
		prd_new.depth = 0;
		prd_new.seed = t;
		prd_new.seed64 = t64;
		Ray new_ray(sample.pos, w_i, radiance_ray_type, scene_epsilon);
		rtTrace(top_object, new_ray, prd_new);
		t = prd_new.seed;
		t64 = prd_new.seed64;
		Le = prd_new.result*M_PIf;
		//cos_theta_i = max(dot(w_i, sample.normal), 0.0f);
	}
	Le /= indirect_prob;
#endif
	cos_theta_i = max(dot(w_i, sample.normal), 0.0f);
	float3 weight = make_float3(1.0f);

//---------------SMOOTH TRANSLUCENT MATERIAL----------------
	if (current_material_type == TRANSLUCENT_SHADER) {
		
		// compute direction of the transmitted lights
		float cos_theta_i_sqr = cos_theta_i*cos_theta_i;
		float sin_theta_t_sqr = recip_ior*recip_ior*(1.0f - cos_theta_i_sqr);
		float cos_theta_t = sqrt(1.0f - sin_theta_t_sqr);
		sample.transmitted = recip_ior*(cos_theta_i*sample.normal - w_i) - sample.normal*cos_theta_t;
		float3 T12 = make_float3(1.0f - fresnel_R(cos_theta_i, cos_theta_t, recip_ior));
		//T12 *= current_scattering_properties.C_phi * 4.0f;
		weight *= T12;
		sample.weight = weight;
		sample.L = Le*(triangles*area);

	} 
//---------------ROUGH TRANSLUCENT MATERIAL----------------
	else if (current_material_type == ROUGH_TRANSLUCENT_SHADER) {

		float a_x = current_roughness.x;
		float a_y = current_roughness.y;
		// compute direction of the transmitted lights
		float cos_theta_i_sqr = cos_theta_i*cos_theta_i;
		float sin_theta_t_sqr = recip_ior*recip_ior*(1.0f - cos_theta_i_sqr);
		float cos_theta_t = sqrt(1.0f - sin_theta_t_sqr);
		sample.transmitted = recip_ior*(cos_theta_i*sample.normal - w_i) - sample.normal*cos_theta_t;
		float3 T12 = make_float3(1.0f - fresnel_R(cos_theta_i, cos_theta_t, recip_ior));
		
		if (current_microfacet_model == WALTER_MODEL) {
			float G_i_m = masking_G1(w_i, sample.normal, n, a_x, a_y, current_normal_distribution);
			float abs_i_m = fabsf(dot(w_i, sample.normal));
			float abs_i_n = fabsf(dot(w_i, n));
			float abs_n_m = fabsf(dot(n, sample.normal));
			float G_o_m = masking_G1(sample.transmitted, sample.normal, n, a_x, a_y, current_normal_distribution);
			weight *= abs_i_m * G_i_m / (abs_i_n * abs_n_m) * G_o_m;
			weight *= T12;
		}
		else if (current_microfacet_model == VISIBLE_NORMALS_MODEL)
		{
			float G_o_m = masking_G1(sample.transmitted, sample.normal, n, a_x, a_y, current_normal_distribution);
			weight *= G_o_m;
			weight *= T12;
		}
		else if (current_microfacet_model == MULTISCATTERING_MODEL) {
			//weight *= microfacet_multiscattering_dielectric_BSDF_eval(w_i, sample.transmitted, n, recip_ior, a_x, a_y, t, 0, current_normal_distribution);
			float G_i_m = masking_G1(w_i, sample.normal, n, a_x, a_y, current_normal_distribution);
			float D = microfacet_eval_visible_normal(w_i, sample.normal, n, a_x, a_y, current_normal_distribution);
			float microfacet_pdf = G_i_m  * D  *fabsf(dot(w_i, sample.normal)) / fabsf(dot(w_i, n));
			weight *=  (T12);

		}
		//rtPrintf("weight %f %f %f \n", weight.x, weight.y, weight.z);
		//weight *= current_scattering_properties.C_phi * 4.0f;	
		sample.weight = weight;
		sample.L = Le*(triangles*area);
	}
}

