#include "hip/hip_runtime.h"
// 02576 OptiX Rendering Framework
// Written by Jeppe Revall Frisvad, 2011
// Copyright (c) DTU Informatics 2011

#define REFLECT
#define TRANSMIT

#define RND_64

#include <optix.h>
#include <optix_math.h>
#include "../helpers.h"
#include "../random.h"
#include "../dipoles/directional_dipole.h"
#include "../dipoles/standard_dipole.h"
#include "../Fresnel.h"
#include "../structs.h"

using namespace optix;

// Standard ray variables
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );
rtDeclareVariable(int, max_depth, , );

// SS properties
rtDeclareVariable(ScatteringMaterialProperties, scattering_properties, , );

// Variables for shading
rtBuffer<PositionSample> samples_output_buffer;
rtDeclareVariable(uint, translucent_index, , );
rtDeclareVariable(uint, samples, , );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(uint, dipole_model, , );

#if defined REFLECT || defined TRANSMIT
// Recursive ray tracing variables
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );
//rtDeclareVariable(unsigned int, radiance_ray_type, , );
#endif

// Any hit program for shadows
RT_PROGRAM void any_hit()
{
	// this material is opaque, so it fully attenuates all shadow rays
	prd_shadow.attenuation = 0.0f;
	rtTerminateRay();
}

// Closest hit program for Lambertian shading using the basic light as a directional source
RT_PROGRAM void closest_hit()
{
	prd_radiance.result = make_float3(0.0f);
	if (prd_radiance.depth > max_depth) return;

	float3 no = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 xo = ray.origin + t_hit*ray.direction;
	float3 wo = -ray.direction;
	ScatteringMaterialProperties& props = scattering_properties;
	float recip_ior = 1.0f / props.relative_ior;
	float3 beam_T = make_float3(1.0f);
	uint& t = prd_radiance.seed;
	Seed64& t64 = prd_radiance.seed64;
	//
#ifdef RND_64
	float reflect_xi = rnd_accurate(t64);
#else
	float reflect_xi = rnd_tea(t);
#endif
	//
#ifdef TRANSMIT
	float cos_theta_o = dot(wo, no);
	bool inside = cos_theta_o < 0.0f;
	if (inside)
	{
		if (dipole_model == DIRECTIONAL_DIPOLE) {
			beam_T = expf(-t_hit*props.deltaEddExtinction);
		}
		else if (dipole_model == STANDARD_DIPOLE) {
			beam_T = expf(-t_hit*props.extinction);
		}
		float prob = (beam_T.x + beam_T.y + beam_T.z) / 3.0f;
#ifdef RND_64
		if (rnd_accurate(t64) >= prob) return;
#else
		if (rnd_tea(t) >= prob) return;
#endif
		beam_T /= prob;
		recip_ior = props.relative_ior;
		no = -no;
		cos_theta_o = -cos_theta_o;
	}
	float sin_theta_t_sqr = recip_ior*recip_ior*(1.0f - cos_theta_o*cos_theta_o);
	float cos_theta_t = 1.0f;
	float R = 1.0f;
	if (sin_theta_t_sqr < 1.0f)
	{
		cos_theta_t = sqrtf(1.0f - sin_theta_t_sqr);
		R = fresnel_R(cos_theta_o, cos_theta_t, recip_ior);
	}
	if (reflect_xi >= R)
	{
		float3 wt = recip_ior*(cos_theta_o*no - wo) - no*cos_theta_t;
		PerRayData_radiance prd_refracted;
		prd_refracted.depth = prd_radiance.depth + 1;
		prd_refracted.seed64 = t64;
		prd_refracted.seed = t;
		prd_refracted.result = make_float3(0.0f);
		prd_refracted.emit_light = 1;
		Ray refracted(xo, wt, radiance_ray_type, scene_epsilon);
		rtTrace(top_object, refracted, prd_refracted);
		prd_radiance.result += prd_refracted.result;
		t = prd_refracted.seed;
		t64 = prd_refracted.seed64;
		if (!inside)
		{
#else
	float cos_theta_o = dot(wo, no);
	float R = fresnel_R(cos_theta_o, recip_ior);
#endif

	//float chosen_transport_rr = props.mean_transport;
	float chosen_transport_rr = fminf(props.transport.x, fminf(props.transport.y, props.transport.z));
	float3 accumulate = make_float3(0.0f);
	uint N = samples_output_buffer.size();
	N = samples;
	for (uint i = 0; i < N; ++i)
	{
		PositionSample& sample = samples_output_buffer[i + translucent_index*N];

		// compute direction of the transmitted light
		const float3& wi = sample.dir;
		float cos_theta_i = max(dot(wi, sample.normal), 0.0f);
		float cos_theta_i_sqr = cos_theta_i*cos_theta_i;
		float sin_theta_t_sqr = recip_ior*recip_ior*(1.0f - cos_theta_i_sqr);
		float cos_theta_t = sqrt(1.0f - sin_theta_t_sqr);
	    //float3 w12 = recip_ior*(cos_theta_i*sample.normal - wi) - sample.normal*cos_theta_t;
		//float T12 = 1.0f - fresnel_R(cos_theta_i, cos_theta_t, recip_ior);
		float3 T12 = sample.weight;
		float3 w12 = sample.transmitted;
		// compute contribution if sample is non-zero
		if (dot(sample.L, sample.L) > 0.0f)
		{
			// Russian roulette
			float dist = length(xo - sample.pos);
			float exp_term = exp(-dist * chosen_transport_rr);
			//exp_term = fmaxf(exp_term, 0.000001f);
#ifdef RND_64
			float rnd_number = rnd_accurate(t64);
#else
			float rnd_number = rnd_tea(t);
#endif
			if (rnd_number < exp_term )
			{
				
				if (dipole_model == DIRECTIONAL_DIPOLE) {
					accumulate += T12*sample.L*dirpole_bssrdf(sample.pos, sample.normal, w12, xo, no, props) / exp_term;
				}
				else if (dipole_model == STANDARD_DIPOLE) {
					accumulate += T12*sample.L*dipole_bssrdf(dist, props) / exp_term;
				}
			}
			else {
				//rtPrintf("no dipole \n");
			}
		}
	}
#ifdef TRANSMIT
	prd_radiance.result += accumulate*props.global_coeff / (float)N;
		}
	}
#else
	float T21 = 1.0f - R;
	prd_radiance.result += T21*accumulate*props.global_coeff / (float)N;
#endif
#ifdef REFLECT
	// Trace reflected ray
	if (reflect_xi < R)
	{
		float3 wr = 2.0f*cos_theta_o*no - wo;
		PerRayData_radiance prd_reflected;
		prd_reflected.depth = prd_radiance.depth + 1;
		prd_reflected.seed = t;
		prd_reflected.seed64 = t64;
		prd_reflected.result = make_float3(0.0f);
		prd_reflected.emit_light = 1;
		Ray reflected(xo, wr, radiance_ray_type, scene_epsilon);
		rtTrace(top_object, reflected, prd_reflected);
		prd_radiance.result += prd_reflected.result;
		t = prd_reflected.seed;
		t64 = prd_reflected.seed64;
	}
#endif
	prd_radiance.seed = t;
	prd_radiance.seed64 = t64;
	prd_radiance.result *= beam_T;
}
