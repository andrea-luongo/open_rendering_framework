#include "hip/hip_runtime.h"
//written by Andrea Luongo

#include <optix.h>
#include <optix_math.h>
#include "../random.h"
#include "../helpers.h"
#include "../structs.h"
#include "../sampler.h"
#include "../Microfacet.h"
#include "../fresnel.h"
#include "../LightSampler.h"


using namespace optix;

// Standard ray variables
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(int, max_depth, , );

// Variables for shading
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

// Recursive ray tracing variables
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(rtObject, top_shadower, , );
//rtDeclareVariable(unsigned int, shadow_ray_type, , );
//rtDeclareVariable(unsigned int, radiance_ray_type, , );
// Material properties 
rtDeclareVariable(float, ior, , );
rtDeclareVariable(float, roughness, , );
rtDeclareVariable(float3, glass_absorption, , );
rtDeclareVariable(uint, microfacet_model, , );
rtDeclareVariable(uint, normal_distribution, , );

__device__ __inline__ void get_rough_glass_rays(const optix::float3& hit_pos, optix::float3& microfacet_normal, const float& ior1_over_ior2,
	optix::Ray& reflected_ray, optix::Ray& refracted_ray, float& R)
{
	// Compute Fresnel reflectance
	float cos_theta = dot(microfacet_normal, -ray.direction);
	float eta = ior1_over_ior2;
	cos_theta = fabsf(cos_theta);
	float sin_theta_t_sqr = eta*eta*(1.0f - cos_theta*cos_theta);
	float cos_theta_t = sqrtf(1.0f - sin_theta_t_sqr);
	R = sin_theta_t_sqr < 1.0f ? fresnel_R(cos_theta, cos_theta_t, eta) : 1.0f;

	float3 refr_dir = eta*ray.direction + microfacet_normal*(eta*cos_theta - cos_theta_t);
	refracted_ray = optix::make_Ray(hit_pos, refr_dir, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);

	float3 reflected_dir = reflect(ray.direction, microfacet_normal);
	reflected_ray = optix::make_Ray(hit_pos, reflected_dir, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
}

// Any hit program for shadows
RT_PROGRAM void any_hit()
{
	// this material is opaque, so it fully attenuates all shadow rays
	prd_shadow.attenuation = 0.0f;
	rtTerminateRay();
}


RT_PROGRAM void closest_hit()
{
	if (prd_radiance.depth >= max_depth)
	{
		prd_radiance.result = make_float3(0.0f);
		return;
	}
	float3 result = make_float3(0.0f);

	// Compute cosine to angle of incidence
	float3 hit_point = ray.origin + t_hit * ray.direction;
	float3 normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 ffnormal = faceforward(normal, -ray.direction, normal);
	float3 w_i = -ray.direction;
	uint& t = prd_radiance.seed;
	optix::Ray refracted_ray, reflected_ray;
	
	float3 beam_T = make_float3(1.0f);
	float cos_theta = dot(-ray.direction, normal);
	if (cos_theta < 0.0f)
	{
		beam_T = expf(-t_hit*glass_absorption);
		float prob = (beam_T.x + beam_T.y + beam_T.z) / 3.0f;
		if (rnd_tea(t) >= prob) return;
		beam_T /= prob;
	}

	float z1 = rnd_tea(t);
	float z2 = rnd_tea(t);
	float3 microfacet_normal;
	float microfacet_D;
	float G_i_m;
	float G_o_m_refl;
	float G_o_m_refr;
	if (microfacet_model == MULTISCATTERING_MODEL) 
	{
		float eta = 1.0f / ior;
		float a_x = roughness;
		float a_y = roughness;
		uint scatteringOrder = 0;
		float3 w_o;
		float3 weight = make_float3(1.0f);

		microfacet_multiscattering_dielectric_BSDF_sample(w_i, w_o, normal, eta, a_x, a_y, t, scatteringOrder, weight, normal_distribution);

		PerRayData_radiance prd_new_ray;
		prd_new_ray.depth = prd_radiance.depth + 1;
		prd_new_ray.result = make_float3(0.0f);
		prd_new_ray.seed = t;
		prd_new_ray.emit_light = 1;
		optix::Ray new_ray = optix::make_Ray(hit_point, w_o, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
		rtTrace(top_object, new_ray, prd_new_ray);
		prd_radiance.seed = prd_new_ray.seed;
		result += prd_new_ray.result * weight;
	}
	else
	{
		if (microfacet_model == WALTER_MODEL)
		{
			microfacet_sample_normal(ffnormal, microfacet_normal, z1, z2, roughness, normal_distribution);
		}
		else if (microfacet_model == VISIBLE_NORMALS_MODEL)
		{
			microfacet_sample_visible_normal(w_i, ffnormal, microfacet_normal, roughness, roughness, z1, z2, normal_distribution);
		}
		// Compute relative index of refraction
		float cos_theta_in = dot(normal, w_i);
		float ior1_over_ior2;
		if (cos_theta_in < 0.0f) {
			ior1_over_ior2 = ior;
		}
		else {
			ior1_over_ior2 = 1.0f / ior;
		}

		if (microfacet_model == WALTER_MODEL)
		{
			G_i_m = masking_G1(w_i, microfacet_normal, ffnormal, roughness, normal_distribution);
		}
		else if (microfacet_model == VISIBLE_NORMALS_MODEL)
		{
			G_i_m = masking_G1(w_i, microfacet_normal, ffnormal, roughness, roughness, normal_distribution);
		}

		//stop ray tracing if the microfacet is not visible from direction i
		if (G_i_m <= 0.0f) {
			prd_radiance.result = result;
			return;
		}
		// Compute Fresnel reflectance (R) and reflected and refracted rays
		float R;
		get_rough_glass_rays(hit_point, microfacet_normal, ior1_over_ior2, reflected_ray, refracted_ray, R);
		float abs_i_m = fabsf(dot(w_i, microfacet_normal));
		float abs_i_n = fabsf(dot(w_i, ffnormal));
		float abs_n_m = fabsf(dot(normal, microfacet_normal));
		float russian_roulette_seed = rnd_tea(prd_radiance.seed);
		PerRayData_radiance prd_new_ray;
		prd_new_ray.depth = prd_radiance.depth + 1;
		prd_new_ray.result = make_float3(0.0f);
		prd_new_ray.seed = t;
		prd_new_ray.emit_light = 1;
		float weight = 1.0f;
		//Russian Roulette to choose between reflection and refraction
		if (russian_roulette_seed > R) {
			if (microfacet_model == WALTER_MODEL)
			{
				G_o_m_refr = masking_G1(refracted_ray.direction, microfacet_normal, ffnormal, roughness, normal_distribution);
			}
			else if (microfacet_model == VISIBLE_NORMALS_MODEL)
			{
				G_o_m_refr = masking_G1(refracted_ray.direction, microfacet_normal, ffnormal, roughness, roughness, normal_distribution);
			}

			if (G_o_m_refr <= 0.0f) {
				prd_radiance.result = result;
				return;
			}
			rtTrace(top_object, refracted_ray, prd_new_ray);
			if (microfacet_model == WALTER_MODEL)
			{
				weight = abs_i_m * G_i_m * G_o_m_refr / (abs_i_n * abs_n_m);
			}
			else if (microfacet_model == VISIBLE_NORMALS_MODEL)
			{
				weight = G_o_m_refr;
			}

		}
		else {
			// Trace reflected ray
			if (microfacet_model == WALTER_MODEL)
			{
				G_o_m_refl = masking_G1(reflected_ray.direction, microfacet_normal, ffnormal, roughness, normal_distribution);
			}
			else if (microfacet_model == VISIBLE_NORMALS_MODEL)
			{
				G_o_m_refl = masking_G1(reflected_ray.direction, microfacet_normal, ffnormal, roughness, roughness, normal_distribution);
			}

			if (G_o_m_refl <= 0.0f) {
				prd_radiance.result = result;
				return;
			}
			rtTrace(top_object, reflected_ray, prd_new_ray);
			if (microfacet_model == WALTER_MODEL)
			{
				weight = abs_i_m * G_i_m * G_o_m_refl / (abs_i_n * abs_n_m);
			}
			else if (microfacet_model == VISIBLE_NORMALS_MODEL)
			{
				weight = G_o_m_refl;
			}

		}
		prd_radiance.seed = prd_new_ray.seed;
		result += prd_new_ray.result * weight;
	}

	prd_radiance.result = result*beam_T;
}
