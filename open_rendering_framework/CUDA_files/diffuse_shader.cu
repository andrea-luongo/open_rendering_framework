#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_math.h>
#include "../random.h"
#include "../structs.h"
#include "../sampler.h"
#include "../LightSampler.h"

using namespace optix;

// Standard ray variables
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

// Variables for shading
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(int, max_depth, , );
// Material properties (corresponding to OBJ mtl params)
rtTextureSampler<float4, 2> diffuse_map;
rtDeclareVariable(float3, emissive, , );
rtDeclareVariable(float3, diffuse_color, , );
// Shadow variables
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_shadower, , );

// Recursive ray tracing variables
rtDeclareVariable(rtObject, top_object, , );


// Any hit program for shadows
RT_PROGRAM void any_hit()
{
	// this material is opaque, so it fully attenuates all shadow rays
	prd_shadow.attenuation = 0.0f;
	rtTerminateRay();
}

// Closest hit program for Lambertian shading using the basic light as a directional source.
// This one includes shadows.
RT_PROGRAM void closest_hit()
{

	if (prd_radiance.depth > max_depth) return;

	float3 hit_pos = ray.origin + t_hit * ray.direction;
	float3 normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 ffnormal = faceforward(normal, -ray.direction, normal);
	float3 rho_d = make_float3(tex2D(diffuse_map, texcoord.x, texcoord.y));
	uint& t = prd_radiance.seed;
	// Emission
	float3 result = /*prd_radiance.emit_light ? emissive :*/ make_float3(0.0f);

	uint light_idx = light_buffer.size()*rnd_tea(t);
	float light_pdf = 1.0f / light_buffer.size();
	LightStruct direct_light = light_buffer[light_idx];
	

	float dist;
	float3 radiance;
	float3 w_l = make_float3(0.0f);
	float cos_theta = 0.0f;

	evaluate_direct_illumination(hit_pos, &direct_light, w_l, radiance, dist, prd_radiance.seed);
	radiance /= light_pdf;
	cos_theta = dot(ffnormal, w_l);

	if (cos_theta > 0.0)
	{
		float V = 1.0f;
		PerRayData_shadow shadow_prd;
		shadow_prd.attenuation = 1.0f;
		Ray shadow_ray(hit_pos, w_l, shadow_ray_type, scene_epsilon, dist - scene_epsilon);
		rtTrace(top_shadower, shadow_ray, shadow_prd);
		V = shadow_prd.attenuation;
		float3 Li = V*radiance;
		result += Li*M_1_PIf*cos_theta*diffuse_color;

		prd_radiance.emit_light = 0;
	}

	// Indirect illumination 

	float prob = 1.0f;
	prob = (diffuse_color.x + diffuse_color.y + diffuse_color.z) / 3.0f;
	float xi = rnd_tea(t);
	if (xi < prob)
	{
		float3 new_dir = sample_cosine_weighted(ffnormal, t);

		PerRayData_radiance prd_new;
		prd_new.depth = prd_radiance.depth + 1;
		prd_new.seed = t;
		prd_new.result = make_float3(0.0f);
		prd_new.emit_light = prd_radiance.emit_light;
		Ray new_ray(hit_pos, new_dir, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
		rtTrace(top_object, new_ray, prd_new);
		result += prd_new.result*diffuse_color / prob;

		prd_radiance.seed = prd_new.seed;
	}
	else
		prd_radiance.seed = t;

	prd_radiance.result = result;
}
