#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_math.h>
#include "../random.h"
#include "../structs.h"
#include "../sampler.h"
#include "../LightSampler.h"
#include "../Fresnel.h"

using namespace optix;



// Standard ray variables
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

// Variables for shading
//rtBuffer<LightStruct> light_buffer;
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(int, max_depth, , );

// Material properties 
rtTextureSampler<float4, 2> diffuse_map;
rtDeclareVariable(float3, emissive, , );
rtDeclareVariable(float3, diffuse_color, , );
rtDeclareVariable(float, ior, , );

// Shadow variables
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_shadower, , );
//rtDeclareVariable(unsigned int, shadow_ray_type, , );


// Recursive ray tracing variables
rtDeclareVariable(rtObject, top_object, , );
//rtDeclareVariable(unsigned int, radiance_ray_type, , );

// Any hit program for shadows
RT_PROGRAM void any_hit()
{
	// this material is opaque, so it fully attenuates all shadow rays
	prd_shadow.attenuation = 0.0f;
	rtTerminateRay();
}

// Closest hit program for Lambertian shading using the basic light as a directional source.
// This one includes shadows.
RT_PROGRAM void closest_hit()
{

	if (prd_radiance.depth > max_depth) return;

	float3 hit_pos = ray.origin + t_hit * ray.direction;
	float3 normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 ffnormal = faceforward(normal, -ray.direction, normal);
	// Emission
	float3 result = make_float3(0.0f);
	uint& t = prd_radiance.seed;
	float n1_over_n2 = 1.0f / ior;
	float r_10 = two_C1(ior);
	float t_10 = 1.0f - r_10;


	float cos_theta_i = dot(-ray.direction, ffnormal);
	//// Compute Fresnel reflectance (R) and trace compute reflected and refracted directions
	float R_i = 1.0f;
	float sin_theta_t_sqr = n1_over_n2*n1_over_n2*(1.0f - cos_theta_i*cos_theta_i);
	float cos_theta_t = 0;
	if (sin_theta_t_sqr < 1.0f)
	{
		cos_theta_t = sqrtf(1.0f - sin_theta_t_sqr);
		R_i = fresnel_R(cos_theta_i, cos_theta_t, n1_over_n2);
	}

	float T_01_i = 1.0f - R_i;
	// Direct illumination
	//for (int i = 0; i < light_buffer.size(); ++i)
	//{
	uint light_idx = light_buffer.size()*rnd_tea(t);
	float light_pdf = 1.0f / light_buffer.size();
	LightStruct direct_light = light_buffer[light_idx];
		//LightStruct direct_light = light_buffer[i];
	float dist;
	float3 radiance;
	float3 w_l = make_float3(0.0f);
	float cos_theta_l = 0.0f;

	evaluate_direct_illumination(hit_pos, &direct_light, w_l, radiance, dist, prd_radiance.seed);
	radiance /= light_pdf;
	cos_theta_l = dot(ffnormal, w_l);

	if (cos_theta_l > 0.0)
	{
		float V = 1.0f;
		PerRayData_shadow shadow_prd;
		shadow_prd.attenuation = 1.0f;
		Ray shadow_ray(hit_pos, w_l, shadow_ray_type, scene_epsilon, dist);
		rtTrace(top_shadower, shadow_ray, shadow_prd);
		V = shadow_prd.attenuation;
		float3 Li = V*radiance;
		//// Compute Fresnel reflectance (R) and trace compute reflected and refracted directions
		float R_l = 1.0f;
		float sin_theta_l_t_sqr = n1_over_n2*n1_over_n2*(1.0f - cos_theta_l*cos_theta_l);
		float cos_theta_l_t = 0;
		if (sin_theta_l_t_sqr < 1.0f)
		{
			cos_theta_l_t = sqrtf(1.0f - sin_theta_l_t_sqr);
			R_l = fresnel_R(cos_theta_l, cos_theta_l_t, n1_over_n2);
		}
		float T_01_l = 1.0f - R_l;

		result += Li*M_1_PIf*cos_theta_l * diffuse_color * n1_over_n2 * n1_over_n2 * T_01_l * T_01_i / (1.0f - diffuse_color * r_10);

		prd_radiance.emit_light = 0;
	}
	//}

	// Indirect illumination 
	
	float prob = 0.5f;
	prob = (diffuse_color.x + diffuse_color.y + diffuse_color.z) / 3.0f;
	prob = R_i;
	float xi = rnd_tea(t);
	if (xi > prob)
	{
		float3 diffuse_dir = sample_cosine_weighted(ffnormal, t);
		float cos_theta_r = dot(diffuse_dir, ffnormal);

		// Compute Fresnel reflectance (R) and compute diffuse contribution
		float R_r = 1.0f;
		float sin_theta_r_t_sqr = n1_over_n2*n1_over_n2*(1.0f - cos_theta_r*cos_theta_r);
		float cos_theta_r_t = 0;
		if (sin_theta_r_t_sqr < 1.0f)
		{
			cos_theta_r_t = sqrtf(1.0f - sin_theta_r_t_sqr);
			R_r = fresnel_R(cos_theta_r, cos_theta_r_t, n1_over_n2);
		}
		float T_01_r = 1.0f - R_r;

		PerRayData_radiance prd_diffuse;
		prd_diffuse.depth = prd_radiance.depth + 1;
		prd_diffuse.seed = t;
		prd_diffuse.emit_light = prd_radiance.emit_light;
		prd_diffuse.result = make_float3(0.0f);
		Ray diffuse_ray(hit_pos, diffuse_dir, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
		rtTrace(top_object, diffuse_ray, prd_diffuse);

		result += prd_diffuse.result * diffuse_color * n1_over_n2 * n1_over_n2 *T_01_r  / (1.0f - diffuse_color * r_10);
		prd_radiance.seed = prd_diffuse.seed;
	}
	else {
		//compute reflectance contribution
		float3 refl_dir = reflect(ray.direction, normal);
		float cos_theta_refl = dot(refl_dir, ffnormal);

		PerRayData_radiance prd_refl;
		prd_refl.depth = prd_radiance.depth + 1;
		prd_refl.seed = prd_radiance.seed;
		prd_refl.emit_light = 1;
		prd_refl.result = make_float3(0.0f);
		Ray refl_ray(hit_pos, refl_dir, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
		rtTrace(top_object, refl_ray, prd_refl);
		result += prd_refl.result ;
		prd_radiance.seed = prd_refl.seed;

	}

	//prd_radiance.depth++;
	prd_radiance.result = result;
}
