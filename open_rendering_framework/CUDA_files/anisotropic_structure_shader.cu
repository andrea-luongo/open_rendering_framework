#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_math.h>
#include "../random.h"
#include "../structs.h"
#include "../sampler.h"
#include "../LightSampler.h"
#include "../Fresnel.h"
#include "../AnisotropicStructures.h"
#include "../MyComplex.h"
using namespace optix;



// Standard ray variables
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

// Variables for shading
//rtBuffer<LightStruct> light_buffer;
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(int, max_depth, , );

// Material properties 
rtTextureSampler<float4, 2> diffuse_map;
rtDeclareVariable(float3, emissive, , );
rtDeclareVariable(float3, diffuse_color, , );
rtDeclareVariable(float, ior, , );
rtDeclareVariable(float, pattern_angle, , );
rtDeclareVariable(float, sinusoid_amplitude, , );
rtDeclareVariable(float2, roughness, , );
rtDeclareVariable(float2, sinusoid_wavelengths, , );
rtDeclareVariable(float, ridge_angle, , );
rtDeclareVariable(uint, structure_type, , );
rtDeclareVariable(rtObject, top_object, , );
rtTextureSampler<float4, 2> texture_sampler;
rtDeclareVariable(float3, texcoord, attribute texcoord, );
// Shadow variables
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_shadower, , );

// Any hit program for shadows
RT_PROGRAM void any_hit()
{
	//// this material is opaque, so it fully attenuates all shadow rays
	prd_shadow.attenuation = 0.0f;
	rtTerminateRay();
}



RT_PROGRAM void closest_hit()
{
	if (prd_radiance.depth >= max_depth)
	{
		prd_radiance.result = make_float3(0.0f);
		return;
	}
	
	float3 result = make_float3(0.0f);
	prd_radiance.result = result;

	//hit point variables
	float3 hit_point = ray.origin + t_hit * ray.direction;
	float3 normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 ffnormal = faceforward(normal, -ray.direction, normal);
	float3 w_i = -ray.direction;
	
	uint& seed = prd_radiance.seed;
	float ior1_over_ior2 = 1.0f / ior;


	//used normal
	float3 n = ffnormal;
	float cos_i_n = dot(w_i, n);
	/*Texturized the plane in order to have chessboard pattern with different ridges orientation
	the ridge are going along the v direction*/
	float a_v = roughness.x;
	float a_u = roughness.y;
	float3 u, v;
	uint type = structure_type;
	float r_angle = ridge_angle;
	float p_angle = pattern_angle * M_PIf / 180.0f;
	float2 lambdas = sinusoid_wavelengths;
	if (type == TEXTURE_STRUCTURE)
	{
		float4 texture_data = tex2D(texture_sampler, texcoord.x, texcoord.y);
		
		if (texture_data.x < 0.5f + 0.01f && texture_data.x > 0.5f - 0.01f /*&& texture_data.y == 1.0f && texture_data.z == 1.0f*/) {

			type = RIDGED_STRUCTURE;
			r_angle = texture_data.y * 90.0f;
			p_angle = texture_data.z * 2.0f  * M_PIf ;
			//ridge_create_rotate_onb(n, u, v, p_angle);
		}
		else if (texture_data.x < 0.25f + 0.01f && texture_data.x > 0.25f - 0.01f /*&& texture_data.y == 1.0f && texture_data.z == 1.0f*/) {
			type = SINUSOIDAL_STRUCTURE;
			lambdas.x = texture_data.y ;
			lambdas.y = texture_data.z ;
			//ridge_create_rotate_onb(n, u, v, p_angle);
		}
		//ridge_create_onb(n, u, v, texcoord.x, texcoord.y);
	}


	if (type == RIDGED_STRUCTURE)
	{
		ridge_create_rotate_onb(n, u, v, p_angle);
	
	}
	else if (type == SINUSOIDAL_STRUCTURE)
	{
		ridge_create_rotate_onb(n, u, v, p_angle);
	}

	//rtPrintf("u %f %f %f v %f %f %f \n", u.x, u.y, u.z, v.x, v.y, v.z);
	//prd_radiance.result = (u)*0.5f + make_float3(0.5f);
	//return;
	//rtPrintf("angle %f \n", r_angle);
	if (type == RIDGED_STRUCTURE)
	{

		//indirect light
		float3 w_o = make_float3(0.0f);
		float3 brdf = make_float3(0.0f);
		float3 m = make_float3(0.0f);
		ridge_sample_BRDF(r_angle, u, v, n, w_i, diffuse_color, ior1_over_ior2, a_u, a_v, seed, m, w_o, brdf);

		if (length(brdf) > 0.0f) {
			PerRayData_radiance prd_new_ray;
			prd_new_ray.depth = prd_radiance.depth + 1;
			prd_new_ray.result = make_float3(0.0f);
			prd_new_ray.seed = seed;
			prd_new_ray.emit_light = 1;
			optix::Ray new_ray = optix::make_Ray(hit_point, w_o, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
			rtTrace(top_object, new_ray, prd_new_ray);
			prd_radiance.seed = prd_new_ray.seed;
			//rtPrintf("brdf %f %f %f\n", brdf.x, brdf.y, brdf.z);
			result += prd_new_ray.result * brdf;
		}

		
		//direct light
		uint light_idx = light_buffer.size()*rnd_tea(seed);
		float light_pdf = 1.0f / light_buffer.size();
		LightStruct direct_light = light_buffer[light_idx];
		float dist;
		float3 radiance;
		float3 w_l = make_float3(0.0f);
		float cos_theta = 0.0f;
		evaluate_direct_illumination(hit_point, &direct_light, w_l, radiance, dist, prd_radiance.seed);
		radiance /= light_pdf;
		float cos_n_l = dot(ffnormal, w_l);
		if (cos_n_l > 0.0)
		{
			float V = 1.0f;
			PerRayData_shadow shadow_prd;
			shadow_prd.attenuation = 1.0f;
			Ray shadow_ray(hit_point, w_l, shadow_ray_type, scene_epsilon, dist - scene_epsilon);
			rtTrace(top_shadower, shadow_ray, shadow_prd);
			V = shadow_prd.attenuation;
			if (V > 0.0f) {
				float3 Li = V*radiance;
				//diffuse part

				float3 direct_brdf = make_float3(0.0f);
				
				ridge_eval_BRDF(r_angle, u, v, n, w_i, w_l, diffuse_color, ior1_over_ior2, a_u, a_v, seed, direct_brdf);
				result += direct_brdf * cos_n_l * Li ;

				//rtPrintf("direct_brdf %f %f %f\n", direct_brdf.x, direct_brdf.y, direct_brdf.z);
			}
		
		}
	}
	else if (type == SINUSOIDAL_STRUCTURE)
	{

		//indirect light
		float3 w_o = make_float3(0.0f);
		float3 brdf = make_float3(0.0f);
		float3 m = make_float3(0.0f);
		float A = sinusoid_amplitude;
		//rtPrintf("lambdas %f %f A %f\n", lambdas.x, lambdas.y, A);
		sinusoid_sample_BRDF(lambdas, A, u, v, n, w_i, diffuse_color, ior1_over_ior2, a_u, a_v, seed, m, w_o, brdf);

		if (length(brdf) > 0.0f) {
			PerRayData_radiance prd_new_ray;
			prd_new_ray.depth = prd_radiance.depth + 1;
			prd_new_ray.result = make_float3(0.0f);
			prd_new_ray.seed = seed;
			prd_new_ray.emit_light = 1;
			optix::Ray new_ray = optix::make_Ray(hit_point, w_o, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
			rtTrace(top_object, new_ray, prd_new_ray);
			prd_radiance.seed = prd_new_ray.seed;
			//rtPrintf("brdf %f %f %f\n", brdf.x, brdf.y, brdf.z);
			result += prd_new_ray.result * brdf;
		}


		//direct light
		uint light_idx = light_buffer.size()*rnd_tea(seed);
		float light_pdf = 1.0f / light_buffer.size();
		LightStruct direct_light = light_buffer[light_idx];
		float dist;
		float3 radiance;
		float3 w_l = make_float3(0.0f);
		float cos_theta = 0.0f;
		evaluate_direct_illumination(hit_point, &direct_light, w_l, radiance, dist, prd_radiance.seed);
		radiance /= light_pdf;
		float cos_n_l = dot(ffnormal, w_l);
		if (cos_n_l > 0.0)
		{
			float V = 1.0f;
			PerRayData_shadow shadow_prd;
			shadow_prd.attenuation = 1.0f;
			Ray shadow_ray(hit_point, w_l, shadow_ray_type, scene_epsilon, dist - scene_epsilon);
			rtTrace(top_shadower, shadow_ray, shadow_prd);
			V = shadow_prd.attenuation;
			if (V > 0.0f) {
				float3 Li = V*radiance;
				//diffuse part

				float3 direct_brdf = make_float3(0.0f);

				sinusoid_eval_BRDF(lambdas, A, u, v, n, w_i, w_l, diffuse_color, ior1_over_ior2, a_u, a_v, seed, direct_brdf);
				result += direct_brdf * cos_n_l * Li;

				//rtPrintf("direct_brdf %f %f %f\n", direct_brdf.x, direct_brdf.y, direct_brdf.z);
			}

		}
	}
	else {

		float3 w_o = sample_cosine_weighted(ffnormal, seed);

		float cos_i_n = dot(w_i, ffnormal);
		float cos_o_n = dot(w_o, ffnormal);
		float F_r = fmaxf(0.0f, fresnel_R(fmaxf(0.0f, (dot(w_i, ffnormal))), ior1_over_ior2));
		float diff_weight = 1.0f;
		float F_t_i_m = 1.0f - fresnel_R(fabsf(cos_i_n), ior1_over_ior2);
		float F_t_o_m = 1.0f - fresnel_R(fabsf(cos_o_n), ior1_over_ior2);
		diff_weight *= F_t_i_m * F_t_o_m ;
		float3 brdf =  diff_weight *diffuse_color;

		PerRayData_radiance prd_diffuse;
		prd_diffuse.depth = prd_radiance.depth + 1;
		prd_diffuse.seed = seed;
		prd_diffuse.emit_light = prd_radiance.emit_light;
		prd_diffuse.result = make_float3(0.0f);
		Ray diffuse_ray(hit_point, w_o, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
		rtTrace(top_object, diffuse_ray, prd_diffuse);
		result += prd_diffuse.result * brdf;
		prd_radiance.seed = prd_diffuse.seed;
		//float r_10 = two_C1(ior);
		//float t_10 = 1.0f - r_10;
		//float cos_theta_i = dot(-ray.direction, ffnormal);
		////// Compute Fresnel reflectance (R) and trace compute reflected and refracted directions
		//float R_i = 1.0f;
		//float sin_theta_t_sqr = ior1_over_ior2*ior1_over_ior2*(1.0f - cos_theta_i*cos_theta_i);
		//float cos_theta_t = 0;
		//if (sin_theta_t_sqr < 1.0f)
		//{
		//	cos_theta_t = sqrtf(1.0f - sin_theta_t_sqr);
		//	R_i = fresnel_R(cos_theta_i, cos_theta_t, ior1_over_ior2);
		//}
		//float T_01_i = 1.0f - R_i;
		//// Direct illumination
		////for (int i = 0; i < light_buffer.size(); ++i)
		////{
		//uint light_idx = light_buffer.size()*rnd_tea(seed);
		//float light_pdf = 1.0f / light_buffer.size();
		//LightStruct direct_light = light_buffer[light_idx];
		////LightStruct direct_light = light_buffer[i];
		//float dist;
		//float3 radiance;
		//float3 w_l = make_float3(0.0f);
		//float cos_theta_l = 0.0f;
		//evaluate_direct_illumination(hit_point, &direct_light, w_l, radiance, dist, prd_radiance.seed);
		//radiance /= light_pdf;
		//cos_theta_l = dot(ffnormal, w_l);
		//if (cos_theta_l > 0.0)
		//{
		//	float V = 1.0f;
		//	PerRayData_shadow shadow_prd;
		//	shadow_prd.attenuation = 1.0f;
		//	Ray shadow_ray(hit_point, w_l, shadow_ray_type, scene_epsilon, dist);
		//	rtTrace(top_shadower, shadow_ray, shadow_prd);
		//	V = shadow_prd.attenuation;
		//	float3 Li = V*radiance;
		//	//// Compute Fresnel reflectance (R) and trace compute reflected and refracted directions
		//	float R_l = 1.0f;
		//	float sin_theta_l_t_sqr = ior1_over_ior2*ior1_over_ior2*(1.0f - cos_theta_l*cos_theta_l);
		//	float cos_theta_l_t = 0;
		//	if (sin_theta_l_t_sqr < 1.0f)
		//	{
		//		cos_theta_l_t = sqrtf(1.0f - sin_theta_l_t_sqr);
		//		R_l = fresnel_R(cos_theta_l, cos_theta_l_t, ior1_over_ior2);
		//	}
		//	float T_01_l = 1.0f - R_l;
		//	result += Li*M_1_PIf*cos_theta_l * diffuse_color * ior1_over_ior2 * ior1_over_ior2 * T_01_l * T_01_i / (1.0f - diffuse_color * r_10);
		//	prd_radiance.emit_light = 0;
		//}
		////}
		//// Indirect illumination 
		//float prob = 0.5f;
		//prob = (diffuse_color.x + diffuse_color.y + diffuse_color.z) / 3.0f;
		//prob = R_i;
		//float xi = rnd_tea(seed);
		//xi = 1;
		//if (xi > prob)
		//{
		//	float3 diffuse_dir = sample_cosine_weighted(ffnormal, seed);
		//	float cos_theta_r = dot(diffuse_dir, ffnormal);
		//	// Compute Fresnel reflectance (R) and compute diffuse contribution
		//	float R_r = 1.0f;
		//	float sin_theta_r_t_sqr = ior1_over_ior2*ior1_over_ior2*(1.0f - cos_theta_r*cos_theta_r);
		//	float cos_theta_r_t = 0;
		//	if (sin_theta_r_t_sqr < 1.0f)
		//	{
		//		cos_theta_r_t = sqrtf(1.0f - sin_theta_r_t_sqr);
		//		R_r = fresnel_R(cos_theta_r, cos_theta_r_t, ior1_over_ior2);
		//	}
		//	float T_01_r = 1.0f - R_r;
		//	PerRayData_radiance prd_diffuse;
		//	prd_diffuse.depth = prd_radiance.depth + 1;
		//	prd_diffuse.seed = seed;
		//	prd_diffuse.emit_light = prd_radiance.emit_light;
		//	prd_diffuse.result = make_float3(0.0f);
		//	Ray diffuse_ray(hit_point, diffuse_dir, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
		//	rtTrace(top_object, diffuse_ray, prd_diffuse);
		//	result += prd_diffuse.result * diffuse_color * ior1_over_ior2 * ior1_over_ior2 *T_01_r *(1-R_i)/ (1.0f - diffuse_color * r_10);
		//	prd_radiance.seed = prd_diffuse.seed;
		//}
		//else {
		//	//compute reflectance contribution
		//	float3 refl_dir = reflect(ray.direction, normal);
		//	float cos_theta_refl = dot(refl_dir, ffnormal);
		//	PerRayData_radiance prd_refl;
		//	prd_refl.depth = prd_radiance.depth + 1;
		//	prd_refl.seed = prd_radiance.seed;
		//	prd_refl.emit_light = 1;
		//	prd_refl.result = make_float3(0.0f);
		//	Ray refl_ray(hit_point, refl_dir, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
		//	rtTrace(top_object, refl_ray, prd_refl);
		//	result += prd_refl.result;
		//	prd_radiance.seed = prd_refl.seed;
		//}
	}


		prd_radiance.result = result;
	
}