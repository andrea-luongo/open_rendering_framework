#include "hip/hip_runtime.h"
// 02576 OptiX Rendering Framework
// Written by Jeppe Revall Frisvad, 2011
// Copyright (c) DTU Informatics 2011


#include <optix_world.h>
#include "../structs.h"
#include "../envmap.h"

using namespace optix;

// Standard ray variables
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_depth, prd_depth, rtPayload, );

// Variables for shading
rtDeclareVariable(Ray, ray, rtCurrentRay, );

// Miss program returning background color
RT_PROGRAM void miss()
{
    prd_radiance.result = env_lookup(ray.direction);
}

RT_PROGRAM void depth_miss()
{
	prd_depth.ray_depth = RT_DEFAULT_MAX;
	prd_depth.normal = make_float3(-1.0f);
}