#include "hip/hip_runtime.h"
// 02576 OptiX Rendering Framework
// Written by Jeppe Revall Frisvad, 2011
// Copyright (c) DTU Informatics 2011

#define REFLECT
#define TRANSMIT
#define DIRECTLIGHT
#define RND_64
#include <optix.h>
#include <optix_math.h>
#include "../helpers.h"
#include "../random.h"
#include "../dipoles/directional_dipole.h"
#include "../dipoles/standard_dipole.h"
#include "../Fresnel.h"
#include "../structs.h"
#include "../Microfacet.h"
#include "../LightSampler.h"
using namespace optix;

// Standard ray variables
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );
rtDeclareVariable(int, max_depth, , );

// SS properties
rtDeclareVariable(ScatteringMaterialProperties, scattering_properties, , );

// Variables for shading
rtBuffer<PositionSample> samples_output_buffer;
rtDeclareVariable(uint, translucent_index, , );
rtDeclareVariable(uint, samples, , );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(uint, dipole_model, , );
rtDeclareVariable(uint, normal_distribution, , );
rtDeclareVariable(uint, microfacet_model, , );
rtDeclareVariable(float2, roughness, , );


#if defined REFLECT || defined TRANSMIT
// Recursive ray tracing variables
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );
//rtDeclareVariable(unsigned int, radiance_ray_type, , );
#endif

#if defined DIRECTLIGHT
rtDeclareVariable(rtObject, top_shadower, , );
//rtDeclareVariable(unsigned int, shadow_ray_type, , );
#endif

// Any hit program for shadows
RT_PROGRAM void any_hit()
{
	// this material is opaque, so it fully attenuates all shadow rays
	prd_shadow.attenuation = 0.0f;
	rtTerminateRay();
}

// Closest hit program for Lambertian shading using the basic light as a directional source
RT_PROGRAM void closest_hit()
{
	prd_radiance.result = make_float3(0.0f);
	if (prd_radiance.depth > max_depth) return;
	float3 result = make_float3(0.0f);

	float3 normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 ffnormal = faceforward(normal, -ray.direction, normal);
	float3 hit_pos = ray.origin + t_hit*ray.direction;
	float3 w_i = -ray.direction;

	ScatteringMaterialProperties& props = scattering_properties;
	float recip_ior = 1.0f / props.relative_ior;
	float3 beam_T = make_float3(1.0f);
	uint& t = prd_radiance.seed;
	Seed64& t64 = prd_radiance.seed64;
	//
#ifdef RND_64
	float reflect_xi = rnd_accurate(t64);
	float z1 = rnd_accurate(t64);
	float z2 = rnd_accurate(t64);
#else
	float reflect_xi = rnd_tea(t);
	float z1 = rnd_tea(t);
	float z2 = rnd_tea(t);
#endif

	float3 microfacet_normal;
	float a_x = roughness.x;
	float a_y = roughness.y;
	if (microfacet_model == WALTER_MODEL)
	{
		microfacet_sample_normal(ffnormal, microfacet_normal, a_x, z1, z2, normal_distribution);
	}
	else
	{
		microfacet_sample_visible_normal(w_i, ffnormal, microfacet_normal, a_x, a_y, z1, z2, normal_distribution);
	}
	//microfacet_normal = ffnormal;
	float cos_theta_i = dot(w_i, microfacet_normal);
	bool inside = dot(w_i, normal) < 0.0f;

#ifdef DIRECTLIGHT

#ifdef RND_64
	uint light_idx = light_buffer.size()*rnd_accurate(t64);
#else
	uint light_idx = light_buffer.size()*rnd_tea(t);
#endif

	float light_pdf = 1.0f / light_buffer.size();
	LightStruct direct_light = light_buffer[light_idx];

	float dist;
	float3 light_radiance;
	float3 w_l = make_float3(0.0f);
	float cos_theta_l = 0.0f;
	evaluate_direct_illumination(hit_pos, &direct_light, w_l, light_radiance, dist, t);
	light_radiance /= light_pdf;
	cos_theta_l = dot(ffnormal, w_l);

	if (!inside && cos_theta_l > 0.0)
	{
		float V = 1.0f;
		PerRayData_shadow shadow_prd;
		shadow_prd.attenuation = 1.0f;
		Ray shadow_ray(hit_pos, w_l, shadow_ray_type, scene_epsilon, dist);
		rtTrace(top_shadower, shadow_ray, shadow_prd);
		V = shadow_prd.attenuation;
		if (V > 0.0f)
		{
			float3 Li = V*light_radiance;
			if (microfacet_model == MULTISCATTERING_MODEL)
			{
				MyComplex eta = MyComplex{ recip_ior, 0 };
				MyComplex3 eta3 = { eta, eta, eta };
				float reflected_brdf = 0.0f;
				reflected_brdf = microfacet_multiscattering_dielectric_BSDF_eval(w_i, w_l, ffnormal, recip_ior, a_x, a_y, t, 0, normal_distribution);
				result += Li * reflected_brdf;
				prd_radiance.emit_light = 0;
			}
			else
			{
				//
				float3 refl_normal = normalize(w_i + w_l);
				float cos_theta_refl = dot(refl_normal, w_l);
				//// Compute Fresnel reflectance (R) 
				float R_refl = 1.0f;
				float sin_theta_refl_t_sqr = recip_ior*recip_ior*(1.0f - cos_theta_refl*cos_theta_refl);
				float cos_theta_refl_t = 0;
				if (sin_theta_refl_t_sqr < 1.0f)
				{
					cos_theta_refl_t = sqrtf(1.0f - sin_theta_refl_t_sqr);
					R_refl = fresnel_R(cos_theta_refl, cos_theta_refl_t, recip_ior);
				}

				float G_i_m_refl = masking_G1(w_i, refl_normal, ffnormal, a_x, a_y, normal_distribution);
				float G_o_m_refl = masking_G1(w_l, refl_normal, ffnormal, a_x, a_y, normal_distribution);
				float D_refl = microfacet_eval_visible_normal(w_i, refl_normal, ffnormal, a_x, a_y, normal_distribution);
				float weight_refl = G_i_m_refl * G_o_m_refl * D_refl / (4.0f * fabsf(dot(w_i, ffnormal) * dot(w_l, ffnormal)));
				result += Li* R_refl * weight_refl;
				prd_radiance.emit_light = 1;
			}
		}
	}
#endif
	//
	if (microfacet_model == WALTER_MODEL || microfacet_model == VISIBLE_NORMALS_MODEL)
	{
#ifdef TRANSMIT

		if (inside)
		{
			if (dipole_model == DIRECTIONAL_DIPOLE) {
				beam_T = expf(-t_hit*props.deltaEddExtinction);
			}
			else if (dipole_model == STANDARD_DIPOLE) {
				beam_T = expf(-t_hit*props.extinction);
			}
			float prob = (beam_T.x + beam_T.y + beam_T.z) / 3.0f;
			//

#ifdef RND_64
			if (rnd_accurate(t64) >= prob) return;
#else
			if (rnd_tea(t) >= prob) return;
#endif
			beam_T /= prob;
			recip_ior = props.relative_ior;
			normal = -normal;
			//microfacet_normal = -microfacet_normal;
			//cos_theta_i = -cos_theta_i;
		}

		float sin_theta_t_sqr = recip_ior*recip_ior*(1.0f - cos_theta_i*cos_theta_i);
		float cos_theta_t = 1.0f;
		float R = 1.0f;
		if (sin_theta_t_sqr < 1.0f)
		{
			cos_theta_t = sqrtf(1.0f - sin_theta_t_sqr);
			R = fresnel_R(cos_theta_i, cos_theta_t, recip_ior);
		}
		if (reflect_xi >= R)
		{
			float3 w_t = recip_ior*(cos_theta_i*microfacet_normal - w_i) - microfacet_normal*cos_theta_t;
			PerRayData_radiance prd_refracted;
			prd_refracted.depth = prd_radiance.depth + 1;
			prd_refracted.seed = t;
			prd_refracted.seed64 = t64;
			prd_refracted.result = make_float3(0.0f);
			prd_refracted.emit_light = 1;
			Ray refracted(hit_pos, w_t, radiance_ray_type, scene_epsilon);
			rtTrace(top_object, refracted, prd_refracted);
			float3 weight = make_float3(1.0f);
			if (microfacet_model == WALTER_MODEL) {
				float G_i_m = masking_G1(w_i, microfacet_normal, ffnormal, a_x, normal_distribution);
				float abs_i_m = fabsf(dot(w_i, microfacet_normal));
				float abs_i_n = fabsf(dot(w_i, ffnormal));
				float abs_n_m = fabsf(dot(ffnormal, microfacet_normal));
				float G_o_m_refr = masking_G1(w_t, microfacet_normal, ffnormal, a_x, normal_distribution);
				weight *= abs_i_m * G_i_m *G_o_m_refr / (abs_i_n * abs_n_m);
			}
			else {
				float G_o_m_refr = masking_G1(w_t, microfacet_normal, ffnormal, a_x, a_y, normal_distribution);
				weight *= G_o_m_refr;
			}
			result += prd_refracted.result * weight;
			t = prd_refracted.seed;
			t64 = prd_refracted.seed64;
			if (!inside)
			{
#else
		float cos_theta_i = dot(w_i, microfacet_normal);
		float R = fresnel_R(cos_theta_i, recip_ior);
#endif

		//float chosen_transport_rr = props.mean_transport;
		float chosen_transport_rr = fminf(props.transport.x, fminf(props.transport.y, props.transport.z));
		float3 accumulate = make_float3(0.0f);
		uint N = samples_output_buffer.size();
		N = samples;
		for (uint i = 0; i < N; ++i)
		{
			PositionSample& sample = samples_output_buffer[i + translucent_index*N];
			float3 T12 = sample.weight;
			float3 w12 = sample.transmitted;
			// compute contribution if sample is non-zero
			if (dot(sample.L, sample.L) > 0.0f)
			{
				// Russian roulette
				float dist = length(hit_pos - sample.pos);
				float exp_term = exp(-dist * chosen_transport_rr);
				//exp_term = fmaxf(exp_term, 0.000001f);
#ifdef RND_64
				float rnd_number = rnd_accurate(t64);
#else
				float rnd_number = rnd_tea(t);
#endif
				if (rnd_number < exp_term && rnd_number > 0.001)
				{
					if (dipole_model == DIRECTIONAL_DIPOLE) {
						accumulate += T12*sample.L*dirpole_bssrdf(sample.pos, sample.normal, w12, hit_pos, normal, props) / exp_term * weight;
					}
					else if (dipole_model == STANDARD_DIPOLE) {
						accumulate += T12*sample.L*dipole_bssrdf(dist, props) / exp_term * weight;
					}
				}
				else {
					//rtPrintf("no dipole \n");
				}
			}
		}
#ifdef TRANSMIT
		result += accumulate*props.global_coeff / (float)N;
			}
		}
#else
		float T21 = 1.0f - R;
		prd_radiance.result += T21*accumulate*props.global_coeff / (float)N;
#endif

#ifdef REFLECT
		// Trace reflected ray
		if (reflect_xi < R)
		{
			float3 w_r = 2.0f*cos_theta_i*microfacet_normal - w_i;
			PerRayData_radiance prd_reflected;
			prd_reflected.depth = prd_radiance.depth + 1;
			prd_reflected.seed = t;
			prd_reflected.seed64 = t64;
			prd_reflected.result = make_float3(0.0f);
			prd_reflected.emit_light = prd_radiance.emit_light;
			Ray reflected(hit_pos, w_r, radiance_ray_type, scene_epsilon);
			rtTrace(top_object, reflected, prd_reflected);



			float3 weight = make_float3(1.0f);
			if (microfacet_model == WALTER_MODEL) {
				float G_i_m = masking_G1(w_i, microfacet_normal, ffnormal, a_x, normal_distribution);
				float abs_i_m = fabsf(dot(w_i, microfacet_normal));
				float abs_i_n = fabsf(dot(w_i, ffnormal));
				float abs_n_m = fabsf(dot(ffnormal, microfacet_normal));
				float G_o_m_refl = masking_G1(w_r, microfacet_normal, ffnormal, a_x, normal_distribution);
				weight *= abs_i_m * G_i_m *G_o_m_refl / (abs_i_n * abs_n_m);
			}
			else {
				float G_o_m_refl = masking_G1(w_r, microfacet_normal, ffnormal, a_x, a_y, normal_distribution);
				weight *= G_o_m_refl;
			}

			result += prd_reflected.result * weight;
			t = prd_reflected.seed;
			t64 = prd_reflected.seed64;
		}
#endif



		prd_radiance.seed = t;
		prd_radiance.seed64 = t64;
		prd_radiance.result = result;
		prd_radiance.result *= beam_T;
	}
	else if (microfacet_model == MULTISCATTERING_MODEL)
	{

#ifdef TRANSMIT



		uint scatteringOrder = 0;
		float3 w_o;
		float3 weight = make_float3(1.0f);
		float3 w_m;
		microfacet_multiscattering_dielectric_BSDF_sample(w_i, w_o, normal, 1.0f / props.relative_ior, a_x, a_y, t, scatteringOrder, weight, normal_distribution);

		bool is_refraction = dot(w_o, ffnormal) < 0.0f;

		if (inside)
		{
			if (dipole_model == DIRECTIONAL_DIPOLE) {
				beam_T = expf(-t_hit*props.deltaEddExtinction);
			}
			else if (dipole_model == STANDARD_DIPOLE) {
				beam_T = expf(-t_hit*props.extinction);
			}
			float prob = (beam_T.x + beam_T.y + beam_T.z) / 3.0f;
#ifdef RND_64
			if (rnd_accurate(t64) >= prob) return;
#else
			if (rnd_tea(t) >= prob) return;
#endif
			beam_T /= prob;
			recip_ior = props.relative_ior;
			normal = -normal;
		}

		if (is_refraction)
		{
			PerRayData_radiance prd_new_ray;
			prd_new_ray.depth = prd_radiance.depth + 1;
			prd_new_ray.result = make_float3(0.0f);
			prd_new_ray.seed = t;
			prd_new_ray.emit_light = 1;
			prd_new_ray.seed64 = t64;
			optix::Ray new_ray = optix::make_Ray(hit_pos, w_o, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
			rtTrace(top_object, new_ray, prd_new_ray);
			prd_radiance.seed = prd_new_ray.seed;
			result += prd_new_ray.result * weight;
			t = prd_new_ray.seed;
			t64 = prd_new_ray.seed64;

			if (!inside)
			{
#else
		float cos_theta_i = dot(w_i, microfacet_normal);
		float R = fresnel_R(cos_theta_i, recip_ior);
#endif

		//float chosen_transport_rr = props.mean_transport;
		float chosen_transport_rr = fminf(props.transport.x, fminf(props.transport.y, props.transport.z));
		float3 accumulate = make_float3(0.0f);
		uint N = samples_output_buffer.size();
		N = samples;
		for (uint i = 0; i < N; ++i)
		{
			PositionSample& sample = samples_output_buffer[i + translucent_index*N];
			float3 T12 = sample.weight;
			float3 w12 = sample.transmitted;
			// compute contribution if sample is non-zero
			if (dot(sample.L, sample.L) > 0.0f)
			{
				// Russian roulette
				float dist = length(hit_pos - sample.pos);
				float exp_term = exp(-dist * chosen_transport_rr);
				//exp_term = fmaxf(exp_term, 0.000001f);
#ifdef RND_64
				float rnd_number = rnd_accurate(t64);
#else
				float rnd_number = rnd_tea(t);
#endif
				if (rnd_number < exp_term && rnd_number > 0.001)
				{
					if (dipole_model == DIRECTIONAL_DIPOLE) {
						accumulate += T12*sample.L*dirpole_bssrdf(sample.pos, sample.normal, w12, hit_pos, normal, props) / exp_term * weight;
					}
					else if (dipole_model == STANDARD_DIPOLE) {
						accumulate += T12*sample.L*dipole_bssrdf(dist, props) / exp_term * weight;
					}
				}
				else {
					//rtPrintf("no dipole \n");
				}
			}
		}
#ifdef TRANSMIT
		result += accumulate*props.global_coeff / (float)N;
			}
		}
#else
		float T21 = 1.0f - R;
		prd_radiance.result += T21*accumulate*props.global_coeff / (float)N;
#endif

#ifdef REFLECT
		// Trace reflected ray
		else
		{


			PerRayData_radiance prd_new_ray;
			prd_new_ray.depth = prd_radiance.depth + 1;
			prd_new_ray.result = make_float3(0.0f);
			prd_new_ray.seed = t;
			prd_new_ray.seed64 = t64;
			prd_new_ray.emit_light = 1;
			optix::Ray new_ray = optix::make_Ray(hit_pos, w_o, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
			rtTrace(top_object, new_ray, prd_new_ray);
			prd_radiance.seed = prd_new_ray.seed;
			result += prd_new_ray.result * weight;
			t = prd_new_ray.seed;
			t64 = prd_new_ray.seed64;
		}
#endif

		prd_radiance.seed = t;
		prd_radiance.seed64 = t64;
		prd_radiance.result = result;
		prd_radiance.result *= beam_T;
	}
}

