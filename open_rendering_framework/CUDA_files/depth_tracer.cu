#include "hip/hip_runtime.h"
// 02576 OptiX Rendering Framework
// Written by Jeppe Revall Frisvad, 2011
// Copyright (c) DTU Informatics 2011

#include <optix_world.h>
#include "../structs.h"
#include "../random.h"

using namespace optix;

// Camera variables
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
//
// Ray generation variables
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(uint, frame, , );

// Window variables
rtBuffer<float4, 2> output_buffer;

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );
// Exception and debugging variables
rtDeclareVariable(float3, exception_color, , );

RT_PROGRAM void depth_tracer()
{
	
	float3 normal = make_float3(0.0f);
	float ray_depth = 0.0f;

	PerRayData_depth prd;
	prd.normal = make_float3(0.0f);
	prd.ray_depth = 0.0f;
	prd.seed = tea<16>(launch_dim.x*launch_index.y + launch_index.x, frame);
	
	float2 jitter = make_float2(rnd_tea(prd.seed), rnd_tea(prd.seed));
	float2 ip_coords = (make_float2(launch_index) + jitter) / make_float2(launch_dim) * 2.0f - 1.0f;
	float3 origin = eye;
	float3 direction = normalize(ip_coords.x*U + ip_coords.y*V + W);
	Ray ray(origin, direction, depth_ray_type, scene_epsilon, RT_DEFAULT_MAX);
	rtTrace(top_object, ray, prd);
	ray_depth += prd.ray_depth;
	normal += (1.0f + prd.normal) * 0.5f;

	if (isfinite(ray_depth))
	{
		float4 curr_sum = (frame != 0) ? output_buffer[launch_index] * ((float)frame) : make_float4(0.0f);
		output_buffer[launch_index] = (make_float4(normal, ray_depth) + curr_sum) / ((float)(frame + 1));
		
	}

	/*float4 curr_sum = (frame != 0) ? output_buffer[launch_index] * ((float)frame) : make_float4(0.0f);
	output_buffer[launch_index] = (make_float4(result, 0.0f) + curr_sum) / ((float)(frame + 1));*/
}

RT_PROGRAM void exception()
{
	output_buffer[launch_index] = make_float4(exception_color, 1.0f);

}
