#include "hip/hip_runtime.h"
#include <optix_world.h>
//#include "LightSampler.h"
#include "../structs.h"


using namespace optix;

#define INDIRECT

// Standard ray variables
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );

// Variables for shading
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, radiance, , );

// Shadow variables
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_shadower, , );
//rtDeclareVariable(unsigned int, shadow_ray_type, , );


// Any hit program for shadows
RT_PROGRAM void any_hit()
{
  // this material is opaque, so it fully attenuates all shadow rays
  prd_shadow.attenuation = 0.0f;
  rtTerminateRay();
}

// Closest hit program for Lambertian shading using a triangle mesh as an area source.
// This one includes shadows.
RT_PROGRAM void closest_hit() 
{ 
  float3 hit_pos = ray.origin + t_hit*ray.direction; 
  float3 normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal)); 
  float3 ffnormal = faceforward(normal, -ray.direction, normal); 
  uint& t = prd_radiance.seed;

  
  float3 result = make_float3(0.0f);
  if (dot(normal, -ray.direction) > 0.0f && prd_radiance.emit_light) {
	  result = radiance;
  }
  
  prd_radiance.result = result; 
}
