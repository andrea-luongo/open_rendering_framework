#include "hip/hip_runtime.h"
// 02576 OptiX Rendering Framework
// Written by Jeppe Revall Frisvad, 2011
// Copyright (c) DTU Informatics 2011

#include <optix_world.h>
#include "envmap.h"

using namespace optix;

// Window variables
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );

// Buffers
rtBuffer<float, 2> env_luminance;
rtBuffer<float> marginal_f;

__host__ __device__ __inline__ float luminance_NTSC(optix::float3 & color)
{
  return optix::dot(color, make_float3(0.2989f, 0.5866f, 0.1145f));
}

RT_PROGRAM void env_luminance_camera()
{
  float2 uv = (make_float2(launch_index) + 0.5f)/make_float2(launch_dim);
  float theta = (1.0f - uv.y)*M_PIf;
  float3 texel = make_float3(tex2D(envmap, uv.x, uv.y));
  env_luminance[launch_index] = luminance_NTSC(texel)*sin(theta);
}

RT_PROGRAM void env_marginal_camera()
{
  if(launch_index.x == 0)
  {
    float c_f_sum = 0.0f;
    for(uint i = 0; i < launch_dim.x; ++i)
    {
      uint2 idx = make_uint2(i, launch_index.y);
      c_f_sum += env_luminance[idx];
    }
    marginal_f[launch_index.y] = c_f_sum/launch_dim.x;
  }
}

RT_PROGRAM void env_pdf_camera()
{
  conditional_pdf[launch_index] = env_luminance[launch_index]/marginal_f[launch_index.y];
  float cdf_sum = 0.0f;
  for(uint i = 0; i <= launch_index.x; ++i)
  {
    uint2 idx = make_uint2(i, launch_index.y);
    cdf_sum += env_luminance[idx];
  }
  cdf_sum /= launch_dim.x;
  conditional_cdf[launch_index] = cdf_sum/marginal_f[launch_index.y];
  if(launch_index == launch_dim - 1)
    conditional_cdf[launch_index] = 1.0f;  // handle numerical instability

  if(launch_index.x == 0)
  {
    float m_f_sum = 0.0f;
    for(uint i = 0; i < marginal_f.size(); ++i)
    {
      m_f_sum += marginal_f[i];
      if(i == launch_index.y)
        cdf_sum = m_f_sum;
    }
    m_f_sum /= launch_dim.y;
    cdf_sum /= launch_dim.y;
    marginal_pdf[launch_index.y] = marginal_f[launch_index.y]/m_f_sum;
    marginal_cdf[launch_index.y] = cdf_sum/m_f_sum;
    if(launch_index.y == launch_dim.y - 1)
      marginal_cdf[launch_index.y] = 1.0f; // handle numerical instability
  }
}
